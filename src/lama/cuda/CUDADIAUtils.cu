#include "hip/hip_runtime.h"
/**
 * @file CUDADIAUtils.cu
 *
 * @license
 * Copyright (c) 2009-2013
 * Fraunhofer Institute for Algorithms and Scientific Computing SCAI
 * for Fraunhofer-Gesellschaft
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 * @endlicense
 *
 * @brief Implementation of DIA utilities with CUDA
 * @author Bea Hornef, Thomas Brandes
 * @date 04.07.2012
 * @since 1.0.0
 */

#include <lama/exception/LAMAAssert.hpp>

#include <lama/LAMAInterface.hpp>
#include <lama/LAMAInterfaceRegistry.hpp>

#include <lama/cuda/utils.cu.h>
#include <cudamem/CUDAError.hpp>
#include <lama/cuda/CUDADIAUtils.hpp>
#include <cudamem/CUDAStreamSyncToken.hpp>
#include <lama/cuda/CUDASettings.hpp>
#include <tracing/tracing.hpp>

// thrust
#include <thrust/device_ptr.h>
#include <thrust/sort.h>

#include <common/bind.hpp>

using namespace memory;
using namespace tasking;
using common::getScalarType;

namespace lama
{

    LAMA_LOG_DEF_LOGGER( CUDADIAUtils::logger, "CUDA.DIAUtils" )

    /* --------------------------------------------------------------------------- */

#include <lama/cuda/CUDATexVector.hpp>

    /* --------------------------------------------------------------------------- */

    template<bool useTexture, bool useSharedMemory>
    __inline__ __device__
    int fetchOffset( const int* const offset_d, int[], const int i )
    {
        return offset_d[i];
    }

    template<>
    __inline__ __device__
    int fetchOffset<true, false>( const int* const offset_d, int[], const int i )
    {
        return fetchVectorX<int, true>( offset_d, i );
    }

    template<>
    __inline__ __device__
    int fetchOffset<true, true>( const int* const, int offset_sm[], const int i )
    {
        return offset_sm[i];
    }

    template<>
    __inline__ __device__
    int fetchOffset<false, true>( const int* const, int offset_sm[], const int i )
    {
        return offset_sm[i];
    }

    /* --------------------------------------------------------------------------- */

    template<typename ValueType, bool useTexture, bool useSharedMem>
    __global__ void normal_gemv_kernel(
                    ValueType* result,
                    const ValueType* x,
                    const ValueType* y,
                    const ValueType alpha,
                    const ValueType beta,
                    const ValueType* diagonalValues,
                    const IndexType* offsets_d,
                    const IndexType numRows,
                    const IndexType numColumns,
                    const IndexType numDiagonals )
    {
        extern __shared__ IndexType offsets_sm[];

        if ( useSharedMem )
        {
            int k = threadIdx.x;
            while ( k < numDiagonals )
            {
                offsets_sm[k] = offsets_d[k];
                k += blockDim.x;
            }
            __syncthreads();
        }

        IndexType i = threadId( gridDim, blockIdx, blockDim, threadIdx );

        if ( i < numRows )
        {
            ValueType summand = beta * y[i];

            ValueType temp = 0.0;

            for ( IndexType idiag = 0; idiag < numDiagonals; idiag++ )
            {
                IndexType j = i + fetchOffset<useTexture, useSharedMem>( offsets_d, offsets_sm, idiag );

                if ( j >= 0 && j < numColumns )
                {
                    ValueType val = diagonalValues[ numRows * idiag + i ];
                    temp += val * fetchVectorX<ValueType, useTexture>( x, j );
                }
            }

            result[i] = alpha * temp + summand;
        }
    }

    /* --------------------------------------------------------------------------- */

    template<typename ValueType, bool useTexture, bool useSharedMem>
    __global__ void normal_gemv_kernel_alpha_one_beta_one(
                    ValueType* result,
                    const ValueType* x,
                    const ValueType* y,
                    const ValueType* diagonalValues,
                    const IndexType* offsets_d,
                    const IndexType numRows,
                    const IndexType numColumns,
                    const IndexType numDiagonals )
    {
        extern __shared__ IndexType offsets_sm[];

        if ( useSharedMem )
        {
            int k = threadIdx.x;
            while ( k < numDiagonals )
            {
                offsets_sm[k] = offsets_d[k];
                k += blockDim.x;
            }
            __syncthreads();
        }

        IndexType i = threadId( gridDim, blockIdx, blockDim, threadIdx );

        if ( i < numRows )
        {
            ValueType summand = y[i];

            ValueType temp = 0.0;

            for ( IndexType idiag = 0; idiag < numDiagonals; idiag++ )
            {
                IndexType j = i + fetchOffset<useTexture, useSharedMem>( offsets_d, offsets_sm, idiag );

                if ( j >= 0 && j < numColumns )
                {
                    ValueType val = diagonalValues[ numRows * idiag + i ];
                    temp += val * fetchVectorX<ValueType, useTexture>( x, j );
                }
            }

            result[i] = temp + summand;
        }
    }

    /* --------------------------------------------------------------------------- */

    template<typename ValueType, bool useTexture, bool useSharedMem>
    __global__ void normal_gemv_kernel_alpha_one_beta_zero(
                    ValueType* result,
                    const ValueType* x,
                    const ValueType* y,
                    const ValueType* diagonalValues,
                    const IndexType* offsets_d,
                    const IndexType numRows,
                    const IndexType numColumns,
                    const IndexType numDiagonals )
    {
        extern __shared__ IndexType offsets_sm[];

        if ( useSharedMem )
        {
            int k = threadIdx.x;
            while ( k < numDiagonals )
            {
                offsets_sm[k] = offsets_d[k];
                k += blockDim.x;
            }
            __syncthreads();
        }

        IndexType i = threadId( gridDim, blockIdx, blockDim, threadIdx );

        if ( i < numRows )
        {
            ValueType temp = 0.0;

            for ( IndexType idiag = 0; idiag < numDiagonals; idiag++ )
            {
                IndexType j = i + fetchOffset<useTexture, useSharedMem>( offsets_d, offsets_sm, idiag );

                if ( j >= 0 && j < numColumns )
                {
                    ValueType val = diagonalValues[ numRows * idiag + i ];
                    temp += val * fetchVectorX<ValueType, useTexture>( x, j );
                }
            }

            result[i] = temp;
        }
    }

    /* --------------------------------------------------------------------------- */

    template<typename ValueType, bool useTexture, bool useSharedMem>
    __global__
    void assign_kernel(
                    ValueType* result,
                    const ValueType* y,
                    const IndexType numRows )
    {
        IndexType i = threadId( gridDim, blockIdx, blockDim, threadIdx );

        if ( i < numRows )
        {
            result[i] = y[i];
        }
    }

    /* --------------------------------------------------------------------------- */

    template<typename ValueType, bool useTexture, bool useSharedMem>
    __global__ void normal_gemv_kernel_alpha_one(
                    ValueType* result,
                    const ValueType* x,
                    const ValueType* y,
                    const ValueType beta,
                    const ValueType* diagonalValues,
                    const IndexType* offsets_d,
                    const IndexType numRows,
                    const IndexType numColumns,
                    const IndexType numDiagonals )
    {
        extern __shared__ IndexType offsets_sm[];

        if ( useSharedMem )
        {
            int k = threadIdx.x;
            while ( k < numDiagonals )
            {
                offsets_sm[k] = offsets_d[k];
                k += blockDim.x;
            }
            __syncthreads();
        }

        IndexType i = threadId( gridDim, blockIdx, blockDim, threadIdx );

        if ( i < numRows )
        {
            ValueType summand = 0.0;

            if ( beta != 0.0 )
            {
                summand = beta * y[i];
            }

            ValueType temp = 0.0;

            for ( IndexType idiag = 0; idiag < numDiagonals; idiag++ )
            {
                IndexType j = i + fetchOffset<useTexture, useSharedMem>( offsets_d, offsets_sm, idiag );

                if ( j >= 0 && j < numColumns )
                {
                    ValueType val = diagonalValues[ numRows * idiag + i ];
                    temp += val * fetchVectorX<ValueType, useTexture>( x, j );
                }
            }

            result[i] = temp + summand;
        }
    }

    /* --------------------------------------------------------------------------- */

    template<typename ValueType, bool useTexture, bool useSharedMem>
    __global__ void normal_gemv_kernel_alpha_zero(
                    ValueType* result,
                    const ValueType* x,
                    const ValueType* y,
                    const ValueType beta,
                    const ValueType* diagonalValues,
                    const IndexType* offsets_d,
                    const IndexType numRows,
                    const IndexType numColumns,
                    const IndexType numDiagonals )
    {
        extern __shared__ IndexType offsets_sm[];

        if ( useSharedMem )
        {
            int k = threadIdx.x;
            while ( k < numDiagonals )
            {
                offsets_sm[k] = offsets_d[k];
                k += blockDim.x;
            }
            __syncthreads();
        }

        IndexType i = threadId( gridDim, blockIdx, blockDim, threadIdx );

        if ( i < numRows )
        {
            result[i] = beta * y[i];
        }
    }

    /* --------------------------------------------------------------------------- */

    template<typename ValueType, bool useTexture, bool useSharedMem>
    __global__ void normal_gemv_kernel_beta_one(
                    ValueType* result,
                    const ValueType* x,
                    const ValueType* y,
                    const ValueType alpha,
                    const ValueType* diagonalValues,
                    const IndexType* offsets_d,
                    const IndexType numRows,
                    const IndexType numColumns,
                    const IndexType numDiagonals )
    {
        extern __shared__ IndexType offsets_sm[];

        if ( useSharedMem )
        {
            int k = threadIdx.x;
            while ( k < numDiagonals )
            {
                offsets_sm[k] = offsets_d[k];
                k += blockDim.x;
            }
            __syncthreads();
        }

        IndexType i = threadId( gridDim, blockIdx, blockDim, threadIdx );

        if ( i < numRows )
        {
            ValueType summand = y[i];

            ValueType temp = 0.0;

            for ( IndexType idiag = 0; idiag < numDiagonals; idiag++ )
            {
                IndexType j = i + fetchOffset<useTexture, useSharedMem>( offsets_d, offsets_sm, idiag );

                if ( j >= 0 && j < numColumns )
                {
                    ValueType val = diagonalValues[ numRows * idiag + i ];
                    temp += val * fetchVectorX<ValueType, useTexture>( x, j );
                }
            }

            result[i] = alpha * temp + summand;
        }
    }

    /* --------------------------------------------------------------------------- */

    template<typename ValueType, bool useTexture, bool useSharedMem>
    __global__ void normal_gemv_kernel_beta_zero(
                    ValueType* result,
                    const ValueType* x,
                    const ValueType* y,
                    const ValueType alpha,
                    const ValueType* diagonalValues,
                    const IndexType* offsets_d,
                    const IndexType numRows,
                    const IndexType numColumns,
                    const IndexType numDiagonals )
    {
        extern __shared__ IndexType offsets_sm[];

        if ( useSharedMem )
        {
            int k = threadIdx.x;
            while ( k < numDiagonals )
            {
                offsets_sm[k] = offsets_d[k];
                k += blockDim.x;
            }
            __syncthreads();
        }

        IndexType i = threadId( gridDim, blockIdx, blockDim, threadIdx );

        if ( i < numRows )
        {
            ValueType temp = 0.0;

            for ( IndexType idiag = 0; idiag < numDiagonals; idiag++ )
            {
                IndexType j = i + fetchOffset<useTexture, useSharedMem>( offsets_d, offsets_sm, idiag );

                if ( j >= 0 && j < numColumns )
                {
                    ValueType val = diagonalValues[ numRows * idiag + i ];
                    temp += val * fetchVectorX<ValueType, useTexture>( x, j );
                }
            }

            result[i] = alpha * temp;
        }
    }

    /* --------------------------------------------------------------------------- */

    template<typename ValueType>
    void CUDADIAUtils::normalGEMV(
                    ValueType result[],
                    const ValueType alpha,
                    const ValueType x[],
                    const ValueType beta,
                    const ValueType y[],
                    const IndexType numRows,
                    const IndexType numColumns,
                    const IndexType numDiagonals,
                    const IndexType diaOffsets[],
                    const ValueType diaValues[],
                    SyncToken* syncToken )
    {
        LAMA_REGION( "CUDA.DIA.normalGEMV" )

        LAMA_LOG_INFO( logger, "normalGEMV<" << getScalarType<ValueType>() << ">"
                        << " result[ " << numRows << "] = " << alpha
                        << " * A( #diags = " << numDiagonals << " ) * x + " << beta << " * y " )

        const IndexType blockSize = CUDASettings::getBlockSize();
        dim3 dimBlock( blockSize, 1, 1 );
        dim3 dimGrid = makeGrid( numRows, dimBlock.x );

        LAMA_CHECK_CUDA_ACCESS

        hipStream_t stream = 0;

        if ( syncToken )
        {
            CUDAStreamSyncToken* cudaStreamSyncToken = dynamic_cast<CUDAStreamSyncToken*>( syncToken );
            LAMA_ASSERT_DEBUG( cudaStreamSyncToken, "no cuda stream sync token provided" )
            stream = cudaStreamSyncToken->getCUDAStream();
        }

        const bool useSharedMem = CUDASettings::useSharedMem();
        const bool useTexture = CUDASettings::useTexture();

        LAMA_LOG_INFO( logger, "Start normal_gemv_kernel<" << getScalarType<ValueType>()
                        << "> <<< blockSize = " << blockSize << ", stream = " << stream
                        << ", useTexture = " << useTexture << ", useSharedMem = " << useSharedMem << ">>>" );

        int sharedMemSize = 0;
        if ( useSharedMem )
        {
            sharedMemSize = numDiagonals * sizeof(int);
        }

        if ( useTexture )
        {
            vectorBindTexture( x );

            if ( !useSharedMem )
            {
                vectorBindTexture( diaOffsets );
            }

            if( useSharedMem )
            {
                if( alpha == 1 && beta == 1 )
                {
                    normal_gemv_kernel_alpha_one_beta_one<ValueType, true, true><<< dimGrid, dimBlock, sharedMemSize, stream >>>(
                                    result, x, y, diaValues, diaOffsets, numRows, numColumns, numDiagonals );
                }
                else if ( alpha == 1 && beta == 0 )
                {
                    normal_gemv_kernel_alpha_one_beta_zero<ValueType, true, true><<< dimGrid, dimBlock, sharedMemSize, stream >>>(
                                    result, x, y, diaValues, diaOffsets, numRows, numColumns, numDiagonals );
                }
                else if ( alpha == 0 && beta == 1 )
                {
                    assign_kernel<ValueType, true, true><<< dimGrid, dimBlock, sharedMemSize, stream >>>(
                                    result, y, numRows );
                }
                else if ( alpha == 1 )
                {
                    normal_gemv_kernel_alpha_one<ValueType, true, true><<< dimGrid, dimBlock, sharedMemSize, stream >>>(
                                    result, x, y, beta, diaValues, diaOffsets, numRows, numColumns, numDiagonals );
                }
                else if ( alpha == 0 )
                {
                    normal_gemv_kernel_alpha_zero<ValueType, true, true><<< dimGrid, dimBlock, sharedMemSize, stream >>>(
                                    result, x, y, beta, diaValues, diaOffsets, numRows, numColumns, numDiagonals );
                }
                else if ( beta == 1 )
                {
                    normal_gemv_kernel_beta_one<ValueType, true, true><<< dimGrid, dimBlock, sharedMemSize, stream >>>(
                                    result, x, y, alpha, diaValues, diaOffsets, numRows, numColumns, numDiagonals );
                }
                else if ( beta == 0 )
                {
                    normal_gemv_kernel_beta_zero<ValueType, true, true><<< dimGrid, dimBlock, sharedMemSize, stream >>>(
                                    result, x, y, alpha, diaValues, diaOffsets, numRows, numColumns, numDiagonals );
                }
                else
                {
                    normal_gemv_kernel<ValueType, true, true><<< dimGrid, dimBlock, sharedMemSize, stream >>>(
                                    result, x, y, alpha, beta, diaValues, diaOffsets, numRows, numColumns, numDiagonals );
                }
            }
            else
            {
                if( alpha == 1 && beta == 1 )
                {
                    normal_gemv_kernel_alpha_one_beta_one<ValueType, true, false><<< dimGrid, dimBlock, sharedMemSize, stream >>>(
                                    result, x, y, diaValues, diaOffsets, numRows, numColumns, numDiagonals );
                }
                else if ( alpha == 1 && beta == 0 )
                {
                    normal_gemv_kernel_alpha_one_beta_zero<ValueType, true, false><<< dimGrid, dimBlock, sharedMemSize, stream >>>(
                                    result, x, y, diaValues, diaOffsets, numRows, numColumns, numDiagonals );
                }
                else if ( alpha == 0 && beta == 1 )
                {
                    assign_kernel<ValueType, true, false><<< dimGrid, dimBlock, sharedMemSize, stream >>>(
                                    result, y, numRows );
                }
                else if ( alpha == 1 )
                {
                    normal_gemv_kernel_alpha_one<ValueType, true, false><<< dimGrid, dimBlock, sharedMemSize, stream >>>(
                                    result, x, y, beta, diaValues, diaOffsets, numRows, numColumns, numDiagonals );
                }
                else if ( alpha == 0 )
                {
                    normal_gemv_kernel_alpha_zero<ValueType, true, false><<< dimGrid, dimBlock, sharedMemSize, stream >>>(
                                    result, x, y, beta, diaValues, diaOffsets, numRows, numColumns, numDiagonals );
                }
                else if ( beta == 1 )
                {
                    normal_gemv_kernel_beta_one<ValueType, true, false><<< dimGrid, dimBlock, sharedMemSize, stream >>>(
                                    result, x, y, alpha, diaValues, diaOffsets, numRows, numColumns, numDiagonals );
                }
                else if ( beta == 0 )
                {
                    normal_gemv_kernel_beta_zero<ValueType, true, false><<< dimGrid, dimBlock, sharedMemSize, stream >>>(
                                    result, x, y, alpha, diaValues, diaOffsets, numRows, numColumns, numDiagonals );
                }
                else
                {
                    normal_gemv_kernel<ValueType, true, false><<< dimGrid, dimBlock, sharedMemSize, stream >>>(
                                    result, x, y, alpha, beta, diaValues, diaOffsets, numRows, numColumns, numDiagonals );
                }
            }
        }
        else
        {
            if( useSharedMem )
            {
                if( alpha == 1 && beta == 1 )
                {
                    normal_gemv_kernel_alpha_one_beta_one<ValueType, false, true><<< dimGrid, dimBlock, sharedMemSize, stream >>>(
                                    result, x, y, diaValues, diaOffsets, numRows, numColumns, numDiagonals );
                }
                else if ( alpha == 1 && beta == 0 )
                {
                    normal_gemv_kernel_alpha_one_beta_zero<ValueType, false, true><<< dimGrid, dimBlock, sharedMemSize, stream >>>(
                                    result, x, y, diaValues, diaOffsets, numRows, numColumns, numDiagonals );
                }
                else if ( alpha == 0 && beta == 1 )
                {
                    assign_kernel<ValueType, false, true><<< dimGrid, dimBlock, sharedMemSize, stream >>>(
                                    result, y, numRows );
                }
                else if ( alpha == 1 )
                {
                    normal_gemv_kernel_alpha_one<ValueType, false, true><<< dimGrid, dimBlock, sharedMemSize, stream >>>(
                                    result, x, y, beta, diaValues, diaOffsets, numRows, numColumns, numDiagonals );
                }
                else if ( alpha == 0 )
                {
                    normal_gemv_kernel_alpha_zero<ValueType, false, true><<< dimGrid, dimBlock, sharedMemSize, stream >>>(
                                    result, x, y, beta, diaValues, diaOffsets, numRows, numColumns, numDiagonals );
                }
                else if ( beta == 1 )
                {
                    normal_gemv_kernel_beta_one<ValueType, false, true><<< dimGrid, dimBlock, sharedMemSize, stream >>>(
                                    result, x, y, alpha, diaValues, diaOffsets, numRows, numColumns, numDiagonals );
                }
                else if ( beta == 0 )
                {
                    normal_gemv_kernel_beta_zero<ValueType, false, true><<< dimGrid, dimBlock, sharedMemSize, stream >>>(
                                    result, x, y, alpha, diaValues, diaOffsets, numRows, numColumns, numDiagonals );
                }
                else
                {
                    normal_gemv_kernel<ValueType, false, true><<< dimGrid, dimBlock, sharedMemSize, stream >>>(
                                    result, x, y, alpha, beta, diaValues, diaOffsets, numRows, numColumns, numDiagonals );
                }
            }
            else
            {
                if( alpha == 1 && beta == 1 )
                {
                    normal_gemv_kernel_alpha_one_beta_one<ValueType, false, false><<< dimGrid, dimBlock, sharedMemSize, stream >>>(
                                    result, x, y, diaValues, diaOffsets, numRows, numColumns, numDiagonals );
                }
                else if ( alpha == 1 && beta == 0 )
                {
                    normal_gemv_kernel_alpha_one_beta_zero<ValueType, false, false><<< dimGrid, dimBlock, sharedMemSize, stream >>>(
                                    result, x, y, diaValues, diaOffsets, numRows, numColumns, numDiagonals );
                }
                else if ( alpha == 0 && beta == 1 )
                {
                    assign_kernel<ValueType, false, false><<< dimGrid, dimBlock, sharedMemSize, stream >>>(
                                    result, y, numRows );
                }
                else if ( alpha == 1 )
                {
                    normal_gemv_kernel_alpha_one<ValueType, false, false><<< dimGrid, dimBlock, sharedMemSize, stream >>>(
                                    result, x, y, beta, diaValues, diaOffsets, numRows, numColumns, numDiagonals );
                }
                else if ( alpha == 0 )
                {
                    normal_gemv_kernel_alpha_zero<ValueType, false, false><<< dimGrid, dimBlock, sharedMemSize, stream >>>(
                                    result, x, y, beta, diaValues, diaOffsets, numRows, numColumns, numDiagonals );
                }
                else if ( beta == 1 )
                {
                    normal_gemv_kernel_beta_one<ValueType, false, false><<< dimGrid, dimBlock, sharedMemSize, stream >>>(
                                    result, x, y, alpha, diaValues, diaOffsets, numRows, numColumns, numDiagonals );
                }
                else if ( beta == 0 )
                {
                    normal_gemv_kernel_beta_zero<ValueType, false, false><<< dimGrid, dimBlock, sharedMemSize, stream >>>(
                                    result, x, y, alpha, diaValues, diaOffsets, numRows, numColumns, numDiagonals );
                }
                else
                {
                    normal_gemv_kernel<ValueType, false, false><<< dimGrid, dimBlock, sharedMemSize, stream >>>(
                                    result, x, y, alpha, beta, diaValues, diaOffsets, numRows, numColumns, numDiagonals );
                }
            }
        }

        if ( !syncToken )
        {
            // synchronize now, unbind used textures

            LAMA_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "normalGEMV for DIA" )

            if ( useTexture )
            {
                vectorUnbindTexture( x );

                if ( !useSharedMem )
                {
                    vectorUnbindTexture( diaOffsets );
                }
            }
        }
        else
        {
            // synchronize by syncToken, delay unbind texture 

            if ( useTexture )
            {
                void ( *unbindV ) ( const ValueType* ) = &vectorUnbindTexture;
                void ( *unbindI ) ( const IndexType* ) = &vectorUnbindTexture;

                syncToken->pushRoutine( boost::bind( unbindV, x ) );

                if ( !useSharedMem )
                {
                    syncToken->pushRoutine( boost::bind( unbindI, diaOffsets ) );
                }
            }
        }
    }

    /* --------------------------------------------------------------------------- */

    template<typename ValueType, bool useTexture, bool useSharedMem>
    __global__ void normal_gevm_kernel(
                    ValueType* result,
                    const ValueType* x,
                    const ValueType* y,
                    const ValueType alpha,
                    const ValueType beta,
                    const ValueType* diagonalValues,
                    const IndexType* offsets_d,
                    const IndexType numRows,
                    const IndexType numColumns,
                    const IndexType numDiagonals )
    {
        extern __shared__ IndexType offsets_sm[];

        if ( useSharedMem )
        {
            int k = threadIdx.x;
            while ( k < numDiagonals )
            {
                offsets_sm[k] = offsets_d[k];
                k += blockDim.x;
            }
            __syncthreads();
        }

        IndexType k = threadId( gridDim, blockIdx, blockDim, threadIdx );

        if ( k < numColumns )
        {
            ValueType summand = beta * y[k];

            ValueType temp = 0.0;

            for ( IndexType ii = 0; ii < numDiagonals; ii++ )
            {
                IndexType i = k - fetchOffset<useTexture, useSharedMem>( offsets_d, offsets_sm, ii );

                if ( i >= 0 && i < numRows )
                {
                    temp += diagonalValues[ numRows * ii + i ] * fetchVectorX<ValueType, useTexture>( x, i );
                }
            }

            result[k] = alpha * temp + summand;
        }
    }

    /* --------------------------------------------------------------------------- */

    template<typename ValueType>
    void CUDADIAUtils::normalGEVM(
                    ValueType result[],
                    const ValueType alpha,
                    const ValueType x[],
                    const ValueType beta,
                    const ValueType y[],
                    const IndexType numRows,
                    const IndexType numColumns,
                    const IndexType numDiagonals,
                    const IndexType diaOffsets[],
                    const ValueType diaValues[],
                    SyncToken* syncToken )
    {
        LAMA_REGION( "CUDA.DIA.normalGEVM" )

        LAMA_LOG_INFO( logger, "normalGEVM<" << getScalarType<ValueType>() << ">"
                        << " result[ " << numRows << "] = " << alpha
                        << " * A( #diags = " << numDiagonals << " ) * x + " << beta << " * y " )

        const IndexType blockSize = CUDASettings::getBlockSize();
        dim3 dimBlock( blockSize, 1, 1 );
        dim3 dimGrid = makeGrid( numColumns, dimBlock.x );

        LAMA_CHECK_CUDA_ACCESS

        hipStream_t stream = 0;

        if ( syncToken )
        {
            CUDAStreamSyncToken* cudaStreamSyncToken = dynamic_cast<CUDAStreamSyncToken*>( syncToken );
            LAMA_ASSERT_DEBUG( cudaStreamSyncToken, "no cuda stream sync token provided" )
            stream = cudaStreamSyncToken->getCUDAStream();
        }

        const bool useSharedMem = CUDASettings::useSharedMem();
        const bool useTexture = CUDASettings::useTexture();

        LAMA_LOG_INFO( logger, "Start normal_gevm_kernel<" << getScalarType<ValueType>()
                        << "> <<< blockSize = " << blockSize << ", stream = " << stream
                        << ", useTexture = " << useTexture << ", useSharedMem = " << useSharedMem << ">>>" );

        int sharedMemSize = 0;
        if ( useSharedMem )
        {
            sharedMemSize = numDiagonals * sizeof(int);
        }

        if ( useTexture )
        {
            vectorBindTexture( x );

            if ( !useSharedMem )
            {
                // @ToDo: be careful, some CUDA devices do not support multiple bind textures, e.g. GeForce 460
                vectorBindTexture( diaOffsets );
            }

            if( useSharedMem )
            {
                normal_gevm_kernel<ValueType, true, true><<< dimGrid, dimBlock, sharedMemSize, stream >>>(
                                result, x, y, alpha, beta, diaValues, diaOffsets, numRows, numColumns, numDiagonals );
            }
            else
            {
                normal_gevm_kernel<ValueType, true, false><<< dimGrid, dimBlock, sharedMemSize, stream >>>(
                                result, x, y, alpha, beta, diaValues, diaOffsets, numRows, numColumns, numDiagonals );
            }
        }
        else
        {
            if( useSharedMem )
            {
                normal_gevm_kernel<ValueType, false, true><<< dimGrid, dimBlock, sharedMemSize, stream >>>(
                                result, x, y, alpha, beta, diaValues, diaOffsets, numRows, numColumns, numDiagonals );
            }
            else
            {
                normal_gevm_kernel<ValueType, false, false><<< dimGrid, dimBlock, sharedMemSize, stream >>>(
                                result, x, y, alpha, beta, diaValues, diaOffsets, numRows, numColumns, numDiagonals );
            }
        }

        if ( !syncToken )
        {
            // synchronize now, unbind used textures

            LAMA_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "normalGEMV for DIA" )

            if ( useTexture )
            {
                vectorUnbindTexture( x );

                if ( !useSharedMem )
                {
                    vectorUnbindTexture( diaOffsets );
                }
            }
        }
        else
        {
            // synchronize by syncToken, delay unbind texture

            if ( useTexture )
            {
                void ( *unbindV ) ( const ValueType* ) = &vectorUnbindTexture;
                void ( *unbindI ) ( const IndexType* ) = &vectorUnbindTexture;

                syncToken->pushRoutine( boost::bind( unbindV, x ) );

                if ( !useSharedMem )
                {
                    syncToken->pushRoutine( boost::bind( unbindI, diaOffsets ) );
                }
            }
        }
    }

    /* --------------------------------------------------------------------------- */

    void CUDADIAUtils::setInterface( DIAUtilsInterface& DIAUtils )
    {
        LAMA_LOG_INFO( logger, "set DIA routines for CUDA in Interface" )

#define LAMA_DIA_UTILS_REGISTER(z, I, _)                                                 \
    LAMA_INTERFACE_REGISTER_T( DIAUtils, normalGEMV, ARITHMETIC_TYPE##I )                \
    LAMA_INTERFACE_REGISTER_T( DIAUtils, normalGEVM, ARITHMETIC_TYPE##I )                \
                                                                                         
        BOOST_PP_REPEAT( ARITHMETIC_TYPE_CNT, LAMA_DIA_UTILS_REGISTER, _ )

#undef LAMA_DIA_UTILS_REGISTER

    }

    /* --------------------------------------------------------------------------- */
    /*    Static registration of the Utils routines                                */
    /* --------------------------------------------------------------------------- */

    bool CUDADIAUtils::registerInterface()
    {
        LAMAInterface& interface = LAMAInterfaceRegistry::getRegistry().modifyInterface( context::CUDA );
        setInterface( interface.DIAUtils );
        return true;
    }

    /* --------------------------------------------------------------------------- */
    /*    Static initialiazion at program start                                    */
    /* --------------------------------------------------------------------------- */

    bool CUDADIAUtils::initialized = registerInterface();

} // namespace lama
