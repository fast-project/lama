#include "hip/hip_runtime.h"
/**
 * @file CUDAELLUtils.cu
 *
 * @license
 * Copyright (c) 2009-2013
 * Fraunhofer Institute for Algorithms and Scientific Computing SCAI
 * for Fraunhofer-Gesellschaft
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 * @endlicense
 *
 * @brief Implementation of ELL utilities with CUDA
 * @author Thomas Brandes
 * @date 04.07.2012
 * @since 1.0.0
 */

// lama cuda
#include <lama/cuda/utils.cu.h>
#include <lama/cuda/CUDAStreamSyncToken.hpp>
#include <lama/cuda/CUDAError.hpp>
#include <lama/cuda/CUDAELLUtils.hpp>
#include <lama/cuda/CUDAUtils.hpp>
#include <lama/cuda/CUDASettings.hpp>

// others
#include <lama/LAMAInterface.hpp>
#include <lama/LAMAInterfaceRegistry.hpp>
#include <lama/macros/unused.hpp>
#include <lama/tracing.hpp>

// cuda
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// thrust
#include <thrust/copy.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/fill.h>
#include <thrust/functional.h>
#include <thrust/gather.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/scatter.h>
#include <thrust/tuple.h>
#include <thrust/transform.h>
#include <thrust/transform_reduce.h>

#include <boost/bind.hpp>

namespace lama
{

LAMA_LOG_DEF_LOGGER( CUDAELLUtils::logger, "CUDA.ELLUtils" )

/* ------------------------------------------------------------------------------------------------------------------ */

texture<float, 1> texELLVectorSXref;

texture<int2, 1> texELLVectorDXref;

texture<int, 1> texELLVectorIref;

__inline__ void vectorELLBindTexture( const float* vector )
{
    LAMA_CUDA_RT_CALL( hipBindTexture( NULL, texELLVectorSXref, vector ), "bind float vector x to texture" )
}

__inline__ void vectorELLBindTexture( const double* vector )
{
    LAMA_CUDA_RT_CALL( hipBindTexture( NULL, texELLVectorDXref, vector ), "bind double vector x to texture" )
}

__inline__ void vectorELLBindTexture( const int* vector )
{
    LAMA_CUDA_RT_CALL( hipBindTexture( NULL, texELLVectorIref, vector ), "bind int vector x to texture" )
}

__inline__ void vectorELLUnbindTexture( const float* )
{
    LAMA_CUDA_RT_CALL( hipUnbindTexture( texELLVectorSXref ), "unbind float vector x from texture" )
}

__inline__ void vectorELLUnbindTexture( const double* )
{
    LAMA_CUDA_RT_CALL( hipUnbindTexture( texELLVectorDXref ), "unbind double vector x from texture" )
}

__inline__ void vectorELLUnbindTexture( const int* )
{
    LAMA_CUDA_RT_CALL( hipUnbindTexture( texELLVectorIref ), "unbind int vector x from texture" )
}

template<typename ValueType, bool useTexture>
__inline__ __device__ 
ValueType fetchELLVectorX( const ValueType* const x, const int i )
{
    return x[i];
}

template<>
__inline__ __device__
float fetchELLVectorX<float, true>( const float* const, const int i )
{
    return tex1Dfetch( texELLVectorSXref, i );
}

template<>
__inline__ __device__
double fetchELLVectorX<double, true>( const double* const, const int i )
{
    int2 v = tex1Dfetch( texELLVectorDXref, i );
    return __hiloint2double( v.y, v.x );
}

template<>
__inline__ __device__
int fetchELLVectorX<int, true>( const int* const, const int i )
{
    return tex1Dfetch( texELLVectorIref, i );
}

/* ------------------------------------------------------------------------------------------------------------------ */
/*                                                  thrust functors                                                   */
/* ------------------------------------------------------------------------------------------------------------------ */

template<typename T>
struct greaterThan
{
    const T x;
    greaterThan( T _x )
        : x( _x )
    {
    }

    __host__ __device__
    T operator()( T y )
    {
        return y > x;
    }
};

template<typename T>
struct notEqual
{
    const T x;
    notEqual( T _x )
        : x( _x )
    {
    }

    __host__ __device__
    T operator()( const IndexType& value, const IndexType& index )
    {
        if ( value > x )
        {
            return index;
        }
        else
        {
            return T( -1 );
        }
    }
};

template<typename T>
struct greaterThanEqual
{
    const T x;
    greaterThanEqual( T _x )
        : x( _x )
    {
    }
    __host__ __device__
    T operator()( T y )
    {
        return y >= x;
    }
};

template<typename T>
struct identity
{
    __host__ __device__
    double operator()( thrust::tuple<T,T> x )
    {
        return thrust::get < 0 > ( x ) == thrust::get < 1 > ( x );
    }
};

template<typename ValueType,typename OtherValueType>
struct multiply
{
    __host__ __device__
    ValueType operator()( ValueType value, OtherValueType otherValue )
    {
        return value * static_cast<ValueType>( otherValue );
    }
};

/* ------------------------------------------------------------------------------------------------------------------ */
/*                                                  countNonEmptyRowsBySizes                                          */
/* ------------------------------------------------------------------------------------------------------------------ */

IndexType CUDAELLUtils::countNonEmptyRowsBySizes( const IndexType sizes[], const IndexType numRows )
{
    LAMA_LOG_INFO( logger, "countNonEmptyRowsBySizes #sizes = " << sizes << " #numRows = " << numRows )

    LAMA_CHECK_CUDA_ACCESS

    thrust::device_ptr<IndexType> sizes_ptr( const_cast<IndexType*>( sizes ) );
    IndexType counter = thrust::transform_reduce( sizes_ptr, sizes_ptr + numRows, greaterThan<IndexType>( 0 ), 0,
                        thrust::plus<IndexType>() );
    return counter;
}

/* ------------------------------------------------------------------------------------------------------------------ */
/*                                                  setNonEmptyRowsBySizes                                            */
/* ------------------------------------------------------------------------------------------------------------------ */

void CUDAELLUtils::setNonEmptyRowsBySizes(
    IndexType rowIndexes[],
    const IndexType numNonEmptyRows,
    const IndexType sizes[],
    const IndexType numRows )
{
    LAMA_LOG_INFO( logger,
                   "setNonEmptyRowsBySizes" << " #rowIndexes = " << rowIndexes << ", #numNonEmptyRows = " << numNonEmptyRows << ", #sizes = " << sizes << ", #numRows = " << numRows )

    LAMA_CHECK_CUDA_ACCESS

    // Create device ptr and help variables
    thrust::device_ptr<IndexType> rowIndexes_ptr( const_cast<IndexType*>( rowIndexes ) );
    thrust::device_ptr<IndexType> sizes_ptr( const_cast<IndexType*>( sizes ) );
    thrust::counting_iterator<IndexType> sequence( 0 );
    thrust::device_vector<IndexType> tmp( numRows );

    // transform array
    thrust::transform( sizes_ptr, sizes_ptr + numRows, sequence, tmp.begin(), notEqual<IndexType>( 0 ) );
    thrust::copy_if( tmp.begin(), tmp.end(), rowIndexes_ptr, greaterThanEqual<IndexType>( 0 ) );
}

/* ------------------------------------------------------------------------------------------------------------------ */
/*                                                  hasDiagonalProperty                                               */
/* ------------------------------------------------------------------------------------------------------------------ */

bool CUDAELLUtils::hasDiagonalProperty( const IndexType numDiagonals, const IndexType ellJA[] )
{
    LAMA_LOG_INFO( logger, "hasDiagonalProperty, #numDiagonals = " << numDiagonals )

    LAMA_CHECK_CUDA_ACCESS

    thrust::device_ptr<IndexType> ellJA_ptr( const_cast<IndexType*>( ellJA ) );
    thrust::counting_iterator<IndexType> sequence( 0 );

    if ( numDiagonals > 0 )
    {
        bool diagonalProperty = thrust::transform_reduce(
                                    thrust::make_zip_iterator( thrust::make_tuple( ellJA_ptr, sequence ) ),
                                    thrust::make_zip_iterator(
                                        thrust::make_tuple( ellJA_ptr + numDiagonals, sequence + numDiagonals ) ),
                                    identity<IndexType>(), true, thrust::logical_and<bool>() );

        return diagonalProperty;
    }
    else
    {
        return false;
    }
}

/* ------------------------------------------------------------------------------------------------------------------ */
/*                                                  check                                                             */
/* ------------------------------------------------------------------------------------------------------------------ */

__global__
void checkKernel(
    const IndexType mNumRows,
    const IndexType mNumValuesPerRow,
    const IndexType mNumColumns,
    const IndexType *ia,
    const IndexType *ja,
    bool *result )
{
    const int i = threadId( gridDim, blockIdx, blockDim, threadIdx );
    if ( i < mNumRows )
    {
        // check ia integrity
        result[i] = ( ia[i] <= mNumValuesPerRow );

        // check ja integrity
        for ( IndexType jj = 0; jj < ia[i]; jj++ )
        {
            IndexType j = ja[jj * mNumRows + i];
            bool jaIntegrity = ( j >= 0 && j < mNumColumns );
            result[i] = result[i] && jaIntegrity;
        }
    }
}

void CUDAELLUtils::check(
    const IndexType numRows,
    const IndexType numValuesPerRow,
    const IndexType numColumns,
    const IndexType *ia,
    const IndexType *ja,
    const char* msg )
{
    LAMA_LOG_INFO( logger,
                   "check # numRows = " << numRows << ", numValuesPerRow = " << numValuesPerRow << ", numColumns = " << numColumns )

    LAMA_CHECK_CUDA_ACCESS

    if ( numRows > 0 )
    {
        thrust::device_ptr<bool> resultPtr = thrust::device_malloc<bool>( numRows );
        thrust::fill( resultPtr, resultPtr + numRows, false );

        bool *resultRawPtr = thrust::raw_pointer_cast( resultPtr );

        const int blockSize = CUDASettings::getBlockSize( numRows );
        dim3 dimBlock( blockSize, 1, 1 );
        dim3 dimGrid = makeGrid( numRows, dimBlock.x );

        checkKernel<<<dimGrid, dimBlock>>>( numRows, numValuesPerRow, numColumns, ia, ja, resultRawPtr );

        LAMA_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "fill result with false failed" )

        bool integrity = thrust::reduce( resultPtr, resultPtr + numRows, true, thrust::logical_and<bool>() );

        LAMA_ASSERT_ERROR( integrity, msg << ": ia to large, or ja out of range" )
    }
    else
    {
        LAMA_ASSERT_EQUAL_ERROR( 0, numValuesPerRow )
        LAMA_ASSERT_EQUAL_ERROR( 0, numColumns )
    }
}

/* ------------------------------------------------------------------------------------------------------------------ */
/*                                                  getRow                                                            */
/* ------------------------------------------------------------------------------------------------------------------ */

template<typename ValueType,typename OtherValueType>
__global__
void getRowKernel(
    OtherValueType *row,
    const IndexType i,
    const IndexType numRows,
    const IndexType numColumns,
    const IndexType rowNumColumns,
    const IndexType *ja,
    const ValueType *values )
{
    const int jj = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( jj < rowNumColumns )
    {
        IndexType pos = jj * numRows + i;
        row[ja[pos]] = static_cast<OtherValueType>( values[pos] );
    }
}

template<typename ValueType,typename OtherValueType>
void CUDAELLUtils::getRow(
    OtherValueType *row,
    const IndexType i,
    const IndexType numRows,
    const IndexType numColumns,
    const IndexType UNUSED( numValuesPerRow ),
    const IndexType *ia,
    const IndexType *ja,
    const ValueType *values )
{
    LAMA_LOG_TRACE( logger, "get row #i = " << i )

    LAMA_CHECK_CUDA_ACCESS

    thrust::device_ptr<OtherValueType> rowPtr( const_cast<OtherValueType*>( row ) );
    thrust::fill( rowPtr, rowPtr + numColumns, 0.0 );

    thrust::device_ptr<IndexType> iaPtr( const_cast<IndexType*>( ia ) );
    thrust::host_vector<IndexType> rowNumColumns( iaPtr + i, iaPtr + i + 1 );

    const int blockSize = CUDASettings::getBlockSize( numRows );
    dim3 dimBlock( blockSize, 1, 1 );
    dim3 dimGrid = makeGrid( rowNumColumns[0], dimBlock.x );

    //TODO: find better CUDA / Thrust implementation
    getRowKernel<<<dimGrid, dimBlock>>>( row, i, numRows, numColumns, rowNumColumns[0], ja, values );

    hipStreamSynchronize( 0 );
    LAMA_CHECK_CUDA_ERROR
}

/* ------------------------------------------------------------------------------------------------------------------ */
/*                                                  getValue                                                          */
/* ------------------------------------------------------------------------------------------------------------------ */

template<typename ValueType>
__global__
void getValueKernel(
    const IndexType i,
    const IndexType j,
    const IndexType numRows,
    const IndexType rowNumColumns,
    const IndexType *ja,
    const ValueType *values,
    ValueType *result )
{
    const int jj = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( jj < rowNumColumns )
    {
        IndexType pos = jj * numRows + i;
        if ( ja[pos] == j )
        {
            result[jj] = values[pos];
        }
    }
}

template<typename ValueType,typename OtherValueType>
OtherValueType CUDAELLUtils::getValue(
    const IndexType i,
    const IndexType j,
    const IndexType numRows,
    const IndexType UNUSED( numValuesPerRow ),
    const IndexType *ia,
    const IndexType *ja,
    const ValueType *values )
{
    LAMA_CHECK_CUDA_ACCESS

    LAMA_LOG_TRACE( logger, "get value i = " << i << ", j = " << j << " numRows = " << numRows )

    thrust::device_ptr<IndexType> iaPtr( const_cast<IndexType*>( ia ) );
    thrust::host_vector<IndexType> rowNumColumnsVec( iaPtr + i, iaPtr + i + 1 );

    IndexType rowNumColumns = rowNumColumnsVec[0];

    if ( rowNumColumns > 0 )
    {
        thrust::device_ptr<ValueType> resultPtr = thrust::device_malloc < ValueType > ( rowNumColumns );
        thrust::fill( resultPtr, resultPtr + rowNumColumns, 0.0 );

        ValueType *resultRawPtr = thrust::raw_pointer_cast( resultPtr );

        const int blockSize = CUDASettings::getBlockSize();
        dim3 dimBlock( blockSize, 1, 1 );
        dim3 dimGrid = makeGrid( rowNumColumns, dimBlock.x );

        getValueKernel<<<dimGrid, dimBlock>>>( i, j, numRows, rowNumColumns, ja, values, resultRawPtr );

        hipStreamSynchronize( 0 );
        LAMA_CHECK_CUDA_ERROR

        return thrust::reduce( resultPtr, resultPtr + rowNumColumns );
    }
    return 0.0;
}

/* ------------------------------------------------------------------------------------------------------------------ */
/*                                                  scaleValue                                                        */
/* ------------------------------------------------------------------------------------------------------------------ */

template<typename ValueType,typename OtherValueType>
void CUDAELLUtils::scaleValue(
    const IndexType numRows,
    const IndexType UNUSED( numValuesPerRow ),
    const IndexType ia[],
    ValueType ellValues[],
    const OtherValueType values[] )
{

    LAMA_LOG_INFO( logger,
                   "scaleValue, #numRows = " << numRows << ", ia = " << ia << ", ellValues = " << ellValues << ", values = " << values )

    LAMA_CHECK_CUDA_ACCESS

    thrust::device_ptr<IndexType> ia_ptr( const_cast<IndexType*>( ia ) );
    thrust::device_ptr<ValueType> ellValues_ptr( const_cast<ValueType*>( ellValues ) );
    thrust::device_ptr<OtherValueType> values_ptr( const_cast<OtherValueType*>( values ) );

    IndexType maxCols = CUDAUtils::maxval( ia, numRows );

    //TODO: maybe find better implementation
    for ( IndexType i = 0; i < maxCols; i++ )
    {
        thrust::transform( ellValues_ptr + i * numRows, ellValues_ptr + i * numRows + numRows, values_ptr,
                           ellValues_ptr + i * numRows, multiply<ValueType,OtherValueType>() );
    }

}

/* ------------------------------------------------------------------------------------------------------------------ */
/*                                                  getCSRValues                                                      */
/* ------------------------------------------------------------------------------------------------------------------ */

template<typename ValueType,typename OtherValueType>
__global__
void ell2csrKernel(
    IndexType *csrJa,
    ValueType *csrValues,
    const IndexType * const csrIa,
    const IndexType numRows,
    const IndexType * const ellIa,
    const IndexType * const ellJa,
    const OtherValueType * const ellValues )
{

    const int i = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( i < numRows )
    {
        IndexType rowSize = ellIa[i];
        IndexType offset = csrIa[i];

        for ( IndexType jj = 0; jj < rowSize; ++jj )
        {
            IndexType pos = jj * numRows + i;
            csrJa[offset + jj] = ellJa[pos];
            csrValues[offset + jj] = static_cast<OtherValueType>( ellValues[pos] );
        }
    }

}

template<typename ELLValueType,typename CSRValueType>
void CUDAELLUtils::getCSRValues(
    IndexType csrJA[],
    CSRValueType csrValues[],
    const IndexType csrIA[],
    const IndexType numRows,
    const IndexType UNUSED( numValuesPerRow ),
    const IndexType ellSizes[],
    const IndexType ellJA[],
    const ELLValueType ellValues[] )
{
    LAMA_REGION( "CUDA.ELL->CSR_values" )

    LAMA_LOG_INFO( logger,
                   "get CSRValues<" << Scalar::getType<ELLValueType>() << ", " << Scalar::getType<CSRValueType>() << ">" << ", #rows = " << numRows )

    LAMA_CHECK_CUDA_ACCESS

    const int blockSize = CUDASettings::getBlockSize();
    dim3 dimBlock( blockSize, 1, 1 );
    dim3 dimGrid = makeGrid( numRows, dimBlock.x );

    //TODO: find better CUDA / Thrust implementation
    ell2csrKernel<<<dimGrid, dimBlock>>>( csrJA, csrValues, csrIA, numRows,
                                          ellSizes, ellJA, ellValues);

    hipStreamSynchronize( 0 );
    LAMA_CHECK_CUDA_ERROR
}

/* ------------------------------------------------------------------------------------------------------------------ */
/*                                                  setCSRValues                                                      */
/* ------------------------------------------------------------------------------------------------------------------ */

template<typename T1,typename T2>
__global__
void csr2ellKernel(
    int* ell_ja,
    T1* ell_values,
    const int* const ell_ia,
    int n,
    int ellNumValuesPerRow,
    const int* const csr_ia,
    const int* const csr_ja,
    const T2* const csr_values )
{
    const int i = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( i < n )
    {
        int ellOffset = i;
        int lastJ = 0;

        for ( int jj = csr_ia[i]; jj < csr_ia[i + 1]; ++jj )
        {
            lastJ = csr_ja[jj];
            ell_ja[ellOffset] = lastJ;
            ell_values[ellOffset] = csr_values[jj];
            ellOffset += n;
        }

        // fill in useful values until length of line

        for ( int jj = ell_ia[i]; jj < ellNumValuesPerRow; ++jj )
        {
            ell_ja[ellOffset] = lastJ;
            ell_values[ellOffset] = 0.0;
            ellOffset += n;
        }
    }
}

template<typename ELLValueType,typename CSRValueType>
void CUDAELLUtils::setCSRValues(
    IndexType ellJA[],
    ELLValueType ellValues[],
    const IndexType ellSizes[],
    const IndexType numRows,
    const IndexType numValuesPerRow,
    const IndexType csrIA[],
    const IndexType csrJA[],
    const CSRValueType csrValues[] )
{
    LAMA_REGION( "CUDA.ELL<-CSR_values" )

    LAMA_LOG_INFO( logger,
                   "set CSRValues<" << Scalar::getType<ELLValueType>() << ", " << Scalar::getType<CSRValueType>() << ">" << ", #rows = " << numRows << ", #values/row = " << numValuesPerRow )

    LAMA_LOG_DEBUG( logger,
                    "ellJA = " << ellJA << ", ellValues = " << ellValues << ", ellSizes = " << ellSizes << ", csrIA = " << csrIA << ", csrJA = " << csrJA << ", csrValues = " << csrValues )

    LAMA_CHECK_CUDA_ACCESS

    const int blockSize = CUDASettings::getBlockSize();
    dim3 dimBlock( blockSize, 1, 1 );
    dim3 dimGrid = makeGrid( numRows, dimBlock.x );

    csr2ellKernel<<<dimGrid, dimBlock>>>( ellJA, ellValues, ellSizes, numRows, numValuesPerRow,
                                          csrIA, csrJA, csrValues);
    
    LAMA_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "csr2ellKernel" );
}

/* ------------------------------------------------------------------------------------------------------------------ */

template<typename ValueType>
__global__
void fillEllKernel(
    int* ell_ja,
    ValueType* ell_values,
    const int* const ell_ia,
    int n,
    int ellNumValuesPerRow )
{
    const int i = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( i < n )
    {
        int lastJ = 0;

        int pos = ell_ia[i];

        int ellOffset = i + pos * n;

        if ( pos > 0  && pos < ellNumValuesPerRow )
        {
            lastJ = ell_ja[ pos - n ];
        }

        // fill in useful values until length of line

        for ( int jj = pos; jj < ellNumValuesPerRow; ++jj )
        {
            ell_ja[ellOffset] = lastJ;
            ell_values[ellOffset] = static_cast<ValueType>( 0 );
            ellOffset += n;
        }
    }
}

template<typename ValueType>
void CUDAELLUtils::fillELLValues(
    IndexType ellJA[],
    ValueType ellValues[],
    const IndexType ellSizes[],
    const IndexType numRows,
    const IndexType numValuesPerRow )
{
    LAMA_LOG_INFO( logger, "fill ELLValues<" << Scalar::getType<ValueType>() )

    LAMA_CHECK_CUDA_ACCESS

    const int blockSize = CUDASettings::getBlockSize();
    dim3 dimBlock( blockSize, 1, 1 );
    dim3 dimGrid = makeGrid( numRows, dimBlock.x );

    fillEllKernel<<<dimGrid, dimBlock>>>( ellJA, ellValues, ellSizes, numRows, numValuesPerRow );

    LAMA_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "fillEllKernel" );
}

/* ------------------------------------------------------------------------------------------------------------------ */
/*    Kernel for  SMV + SV                                                                                            */
/* ------------------------------------------------------------------------------------------------------------------ */

template<typename T,bool useTexture>
__global__
void normal_gemv_kernel(
    T* result,
    const T* const x_d,
    const T* const y_d,
    T alpha,
    const T beta,
    const T* ellValues,
    const int* ellJA,
    int numRows,
    const int * ellIA)
{
    const int i = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( i < numRows )
    {
        T summand = 0.0;
        if ( beta != 0.0 )
        {
            summand = beta * y_d[i];
        }

        T value = 0.0;
        int pos = i;
        for ( int kk = 0; kk < ellIA[i]; ++kk )
        {
            //if (aValue != 0.0) //compute capability >= 2.0  => disadvantage
            value += ellValues[pos] * fetchELLVectorX<T, useTexture>( x_d, ellJA[pos] );
            pos += numRows;
        }
        result[i] = alpha * value + summand;
    }
}

/* ------------------------------------------------------------------------------------------------------------------ */

template<typename ValueType>
void CUDAELLUtils::normalGEMV(
    ValueType result[],
    const ValueType alpha,
    const ValueType x[],
    const ValueType beta,
    const ValueType y[],
    const IndexType numRows,
    const IndexType numNonZerosPerRow,
    const IndexType ellIA[],
    const IndexType ellJA[],
    const ValueType ellValues[],
    SyncToken* syncToken )
{
    LAMA_REGION( "CUDA.ELL.normalGEMV" )

    LAMA_LOG_INFO( logger, "normalGEMV<" << Scalar::getType<ValueType>() << ">" <<
                           " result[ " << numRows << "] = " << alpha << " * A(ell) * x + " << beta << " * y " )

    LAMA_LOG_DEBUG( logger, "x = " << x << ", y = " << y << ", result = " << result )

    LAMA_CHECK_CUDA_ACCESS

    hipStream_t stream = 0;

    if ( syncToken )
    {
        CUDAStreamSyncToken* cudaStreamSyncToken = dynamic_cast<CUDAStreamSyncToken*>( syncToken );
        LAMA_ASSERT_DEBUG( cudaStreamSyncToken, "no cuda stream sync token provided" )
        stream = cudaStreamSyncToken->getCUDAStream();
        LAMA_LOG_INFO( logger, "asyncronous execution on stream " << stream );
    }

    const int blockSize = CUDASettings::getBlockSize();
    dim3 dimBlock( blockSize, 1, 1 );
    dim3 dimGrid = makeGrid( numRows, dimBlock.x );

    bool useTexture = CUDASettings::useTexture();

    LAMA_LOG_INFO( logger, "Start normal_gemv_kernel<" << Scalar::getType<ValueType>()
                           << "> <<< blockSize = " << blockSize << ", stream = " << stream 
                           << ", useTexture = " << useTexture << ">>>" )

    if ( useTexture )
    {
        vectorELLBindTexture( x );

        LAMA_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( normal_gemv_kernel<ValueType), true>, hipFuncCachePreferL1 ),
                           "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )
     
        normal_gemv_kernel<ValueType, true> <<<dimGrid, dimBlock, 0, stream>>> (
                        result, x, y, alpha, beta, ellValues, ellJA, numRows, ellIA );
    }
    else
    {
        LAMA_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( normal_gemv_kernel<ValueType), false>, hipFuncCachePreferL1 ),
                           "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )

        normal_gemv_kernel<ValueType, false> <<<dimGrid, dimBlock, 0, stream>>> (
                        result, x, y, alpha, beta, ellValues, ellJA, numRows, ellIA );
    }

    if ( !syncToken )
    {
        LAMA_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "sync for normal_gemv_kernel failed" )

        if ( useTexture )
        {
            vectorELLUnbindTexture( x );
        }
    }
    else
    { 
        // synchronization at SyncToken, delay unbind

        if ( useTexture )
        {
            void ( *unbind ) ( const ValueType* ) = &vectorELLUnbindTexture;

            syncToken->pushRoutine( boost::bind( unbind, x ) );
        }
    }
}

/* ------------------------------------------------------------------------------------------------------------------ */
/*    Kernel for  SVM + SV                                                                                            */
/* ------------------------------------------------------------------------------------------------------------------ */

template<typename T, bool useTexture>
__global__
void normal_gevm_kernel(
    T* result,
    const T* x_d,
    const T* y_d,
    const T alpha,
    const T beta,
    const T* ellValues,
    const int* ellJA,
    int numRows,
    int numColumns,
    const int* ellIA )
{
    // result = alpha * x_d * A + beta * y_d

    const int i = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( i < numColumns )
    {
        T summand = beta * y_d[i];
        T value = 0.0;

        for( int j = 0; j < numRows; ++j )
        {
            int pos = j;
            for ( int kk = 0; kk < ellIA[j]; ++kk )
            {
                if( ellJA[pos] == i )
                {
                    value += ellValues[pos] * fetchELLVectorX<T, useTexture>( x_d, j );
                }
                pos += numRows;
            }
        }
        result[i] = alpha * value + summand;
    }
}

/* ------------------------------------------------------------------------------------------------------------------ */

template<typename ValueType>
void CUDAELLUtils::normalGEVM(
    ValueType result[],
    const ValueType alpha,
    const ValueType x[],
    const ValueType beta,
    const ValueType y[],
    const IndexType numRows,
    const IndexType numColumns,
    const IndexType numValuesPerRow,
    const IndexType ellSizes[],
    const IndexType ellJA[],
    const ValueType ellValues[],
    SyncToken* syncToken )
{
    LAMA_LOG_INFO( logger, "normalGEVM<" << Scalar::getType<ValueType>() << ">" <<
                           " result[ " << numColumns << "] = " << alpha << " * A(ell) * x + " << beta << " * y " )

    LAMA_LOG_DEBUG( logger, "x = " << x << ", y = " << y << ", result = " << result )

    LAMA_CHECK_CUDA_ACCESS

    hipStream_t stream = 0; // default stream if no syncToken is given

    const int blockSize = CUDASettings::getBlockSize();

    dim3 dimBlock( blockSize, 1, 1 );

    dim3 dimGrid = makeGrid( numColumns, dimBlock.x );

    bool useTexture = CUDASettings::useTexture();

    if ( syncToken )
    {
        CUDAStreamSyncToken* cudaStreamSyncToken = dynamic_cast<CUDAStreamSyncToken*>( syncToken );
        LAMA_ASSERT_DEBUG( cudaStreamSyncToken, "no cuda stream sync token provided" )
        stream = cudaStreamSyncToken->getCUDAStream();
    }

    LAMA_LOG_INFO( logger, "Start normal_gevm_kernel<" << Scalar::getType<ValueType>()
                           << ", useTexture = " << useTexture << ">" );

    if ( useTexture )
    {
        vectorELLBindTexture( x );

        LAMA_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( normal_gevm_kernel<ValueType), true>, hipFuncCachePreferL1 ),
                           "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )

        normal_gevm_kernel<ValueType, true> <<< dimGrid, dimBlock, 0, stream >>>
                    ( result, x, y, alpha, beta, ellValues, ellJA, numRows, numColumns, ellSizes );
    }
    else
    {
        LAMA_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( normal_gevm_kernel<ValueType), false>, hipFuncCachePreferL1 ),
                           "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )

        normal_gevm_kernel<ValueType, false> <<< dimGrid, dimBlock, 0, stream >>>
                    ( result, x, y, alpha, beta, ellValues, ellJA, numRows, numColumns, ellSizes );
    }

    if ( !syncToken )
    {
        LAMA_CUDA_RT_CALL( hipStreamSynchronize( stream ), "normalGEVM, stream = " << stream )
        LAMA_LOG_DEBUG( logger, "normalGEVM<" << Scalar::getType<ValueType>() << "> synchronized" )
    }

    if ( useTexture )
    {
        if ( !syncToken )
        {
            vectorELLUnbindTexture( x );
        }
        else
        {
             // get routine with the right signature
             void ( *unbind ) ( const ValueType* ) = &vectorELLUnbindTexture;

             // delay unbind until synchroniziaton
             syncToken->pushRoutine( boost::bind( unbind, x ) );
        }
    }
}

/* ------------------------------------------------------------------------------------------------------------------ */

template<typename T,bool useTexture>
__global__
void sparse_gemv_kernel(
    T* const result_d,
    const T* const x_d,
    const T alpha,
    const T* const ellValues,
    const int* const ellIA,
    const int* const ellJA,
    const int* const rowIndexes,
    const int numNonZeroRows,
    int numRows,
    int numValuesPerRow )
{
    // each thread is assigned to one non-zero row

    const int id = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( id < numNonZeroRows )
    {
        const int i = rowIndexes[id];

        int pos = i;

        T value = 0.0;

        const int nonZeros = ellIA[i];

        for ( int kk = 0; kk < nonZeros; ++kk )
        {
            const T aValue = ellValues[pos];

            // compute capability >= 2.0: no benefits to mask with value != 0.0

            value += aValue * fetchELLVectorX<T, useTexture>( x_d, ellJA[pos] );
            pos   += numRows;
        }

        result_d[i] += alpha * value;
    }
}

/* ------------------------------------------------------------------------------------------------------------------ */

template<typename ValueType>
void CUDAELLUtils::sparseGEMV(
    ValueType result[],
    const ValueType alpha,
    const ValueType x[],
    const IndexType numRows,
    const IndexType numNonZerosPerRow,
    const IndexType numNonZeroRows,
    const IndexType rowIndexes[],
    const IndexType ellSizes[],
    const IndexType ellJA[],
    const ValueType ellValues[],
    SyncToken* syncToken )
{
    LAMA_REGION( "CUDA.ELL.sparseGEMV" )

    LAMA_LOG_INFO( logger, "sparseGEMV<" << Scalar::getType<ValueType>() << ">" << ", #non-zero rows = " << numNonZeroRows )

    LAMA_CHECK_CUDA_ACCESS

    hipStream_t stream = 0;

    if ( syncToken )
    {
        CUDAStreamSyncToken* cudaStreamSyncToken = dynamic_cast<CUDAStreamSyncToken*>( syncToken );
        LAMA_ASSERT_DEBUG( cudaStreamSyncToken, "no cuda stream sync token provided" )
        stream = cudaStreamSyncToken->getCUDAStream();
    }

    const int blockSize = CUDASettings::getBlockSize( numNonZeroRows );
    dim3 dimBlock( blockSize, 1, 1 );
    dim3 dimGrid = makeGrid( numNonZeroRows, dimBlock.x );

    bool useTexture = CUDASettings::useTexture();

    if ( useTexture )
    {
        vectorELLBindTexture( x );
    }

    LAMA_LOG_INFO( logger, "Start ell_sparse_gemv_kernel<" << Scalar::getType<ValueType>()
                           << "> <<< blockSize = " << blockSize << ", stream = " << stream 
                           << ", useTexture = " << useTexture << ">>>" );

    if ( useTexture )
    {
        LAMA_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( sparse_gemv_kernel<ValueType), true>, hipFuncCachePreferL1),
                           "hipFuncSetCacheConfig failed" )
     
        sparse_gemv_kernel<ValueType, true> <<<dimGrid, dimBlock, 0, stream>>>(
            result, x, alpha, ellValues, ellSizes, ellJA, rowIndexes, numNonZeroRows, numRows, numNonZerosPerRow );
    }
    else
    {
        LAMA_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( sparse_gemv_kernel<ValueType), false>, hipFuncCachePreferL1),
                           "hipFuncSetCacheConfig failed" )
     
        sparse_gemv_kernel<ValueType, false> <<<dimGrid, dimBlock, 0, stream>>>(
            result, x, alpha, ellValues, ellSizes, ellJA, rowIndexes, numNonZeroRows, numRows, numNonZerosPerRow );
    }

    if ( !syncToken )
    {
        LAMA_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "sparse GEMV kernel failed" )

        if ( useTexture )
        {
            vectorELLUnbindTexture( x );
        }
    }
    else
    { 
        // synchronization at SyncToken, delay unbind

        if ( useTexture )
        {
            void ( *unbind ) ( const ValueType* ) = &vectorELLUnbindTexture;

            syncToken->pushRoutine( boost::bind( unbind, x ) );
        }
    }
}

/* ------------------------------------------------------------------------------------------------------------------ */

template<typename T,bool useTexture>
__global__
void sparse_gevm_kernel(
    T* const result_d,
    const T* const x_d,
    const T alpha,
    const T* const ellValues,
    const int* const ellSizes,
    const int* const ellJA,
    const int* const rowIndexes,
    const int numNonZeroRows,
    int numRows,
    int numColumns )
{
    // each thread is assigned to one non-zero row

    const int id = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( id < numColumns )
    {
        T value = 0.0;
        for ( int i = 0; i < numNonZeroRows; ++i )
        {
            int pos = id;

            const int nonZeros = ellSizes[pos];

            for ( int kk = 0; kk < nonZeros; ++kk )
            {
                if( ellJA[pos] == id )
                {
                    const T aValue = ellValues[pos];

                    // compute capability >= 2.0: no benefits to mask with value != 0.0

                    value += aValue * fetchELLVectorX<T, useTexture>( x_d, ellJA[pos] );
                }
                pos   += numRows;
            }
        }
        result_d[id] += alpha * value;
    }
}

/* ------------------------------------------------------------------------------------------------------------------ */

template<typename ValueType>
void CUDAELLUtils::sparseGEVM(
    ValueType result[],
    const ValueType alpha,
    const ValueType x[],
    const IndexType numRows,
    const IndexType numColumns,
    const IndexType numNonZerosPerRow,
    const IndexType numNonZeroRows,
    const IndexType rowIndexes[],
    const IndexType ellSizes[],
    const IndexType ellJA[],
    const ValueType ellValues[],
    SyncToken* syncToken )
{
    LAMA_LOG_INFO( logger,
                   "sparseGEVM<" << Scalar::getType<ValueType>() << ">" << ", #non-zero rows = " << numNonZeroRows )

    LAMA_CHECK_CUDA_ACCESS

    hipStream_t stream = 0;

    if ( syncToken )
    {
        CUDAStreamSyncToken* cudaStreamSyncToken = dynamic_cast<CUDAStreamSyncToken*>( syncToken );
        LAMA_ASSERT_DEBUG( cudaStreamSyncToken, "no cuda stream sync token provided" )
        stream = cudaStreamSyncToken->getCUDAStream();
    }

    const int blockSize = CUDASettings::getBlockSize( numNonZeroRows );

    dim3 dimBlock( blockSize, 1, 1 );

    dim3 dimGrid = makeGrid( numNonZeroRows, dimBlock.x );

    sparse_gevm_kernel<ValueType, false> <<< dimGrid, dimBlock, 0, stream >>>
                    ( result, x, alpha, ellValues, ellSizes, ellJA, rowIndexes, numNonZeroRows, numRows, numColumns );

    if ( !syncToken )
    {
        LAMA_CUDA_RT_CALL( hipStreamSynchronize( stream ), "sparseGEVM, stream = " << stream )
        LAMA_LOG_INFO( logger, "sparseGEVM<" << Scalar::getType<ValueType>() << "> synchronized" )
    }
}

/* ------------------------------------------------------------------------------------------------------------------ */
/*                                                  Jacobi                                                           */
/* ------------------------------------------------------------------------------------------------------------------ */

template<typename T,bool useTexture>
__global__
void ell_jacobi_kernel(
    const int* ellIA,
    const int* ellJA,
    const T* ellValues,
    const int numRows,
    const T* const rhs,
    T* const solution,
    const T* const oldSolution,
    const T omega )
{
    const int i = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( i < numRows )
    {
        T temp = rhs[i];
        ellValues += i;
        ellJA += i;

        const T diag = *ellValues;
        ellValues += numRows;
        ellJA += numRows;

        for ( int kk = 1; kk < ellIA[i]; ++kk )
        {
            const T aValue = *ellValues;
            temp -= aValue * fetchELLVectorX<T, useTexture>( oldSolution, *ellJA );
            ellValues += numRows;
            ellJA += numRows;
        }
        if ( omega == 0.5 )
        {
            solution[i] = omega * ( fetchELLVectorX<T, useTexture>( oldSolution, i ) + temp / diag );
        }
        else if ( omega == 1.0 )
        {
            solution[i] = temp / diag;
        }
        else
        {
            solution[i] = omega * ( temp / diag ) + ( 1.0 - omega ) * fetchELLVectorX<T, useTexture>( oldSolution, i );
        }
    }
}

template<typename ValueType>
void CUDAELLUtils::jacobi(
    ValueType solution[],
    const IndexType numRows,
    const IndexType UNUSED(ellNumValuesPerRow),
    const IndexType* ellSizes,
    const IndexType ellJA[],
    const ValueType ellValues[],
    const ValueType oldSolution[],
    const ValueType rhs[],
    const ValueType omega,
    SyncToken* syncToken )
{
    LAMA_REGION( "CUDA.ELL.jacobi" )

    LAMA_LOG_INFO( logger, "jacobi, #rows = " << numRows )

    LAMA_CHECK_CUDA_ACCESS

    hipStream_t stream = 0;

    const bool useTexture = CUDASettings::useTexture();

    if ( syncToken )
    {
        CUDAStreamSyncToken* cudaStreamSyncToken = dynamic_cast<CUDAStreamSyncToken*>( syncToken );
        LAMA_ASSERT_DEBUG( cudaStreamSyncToken, "no cuda stream sync token provided" )
        stream = cudaStreamSyncToken->getCUDAStream();
    }

    const int blockSize = CUDASettings::getBlockSize( numRows );
    dim3 dimBlock( blockSize, 1, 1 );
    dim3 dimGrid = makeGrid( numRows, dimBlock.x );

    LAMA_LOG_INFO( logger, "Start ell_jacobi_kernel<" << Scalar::getType<ValueType>()
                           << "> <<< block size = " << blockSize << ", stream = " << stream
                           << ", useTexture = " << useTexture << ">>>" );

    if ( useTexture )
    {
        vectorELLBindTexture( oldSolution );

        LAMA_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( ell_jacobi_kernel<ValueType), true>, hipFuncCachePreferL1),
                           "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )

        ell_jacobi_kernel<ValueType, true> <<<dimGrid, dimBlock, 0, stream>>>( ellSizes, ellJA, ellValues,
                numRows, rhs, solution, oldSolution, omega );
    }

    else
    {
        LAMA_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( ell_jacobi_kernel<ValueType), false>, hipFuncCachePreferL1 ),
                           "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )

        ell_jacobi_kernel<ValueType, false> <<<dimGrid, dimBlock, 0, stream>>>( ellSizes, ellJA, ellValues,
                numRows, rhs, solution, oldSolution, omega );
    }

    LAMA_CUDA_RT_CALL( hipGetLastError(), "LAMA_STATUS_DCSRJACOBI_CUDAKERNEL_FAILED" )

    if ( !syncToken )
    {
        // synchronize now and unbind texture if used

        LAMA_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "ELL: jacobiKernel FAILED" )

        if ( useTexture )
        {
            vectorELLUnbindTexture( oldSolution );
        }
    }
    else
    {
        if ( useTexture )
        {
            void ( *unbind ) ( const ValueType* ) = &vectorELLUnbindTexture;

            syncToken->pushRoutine( boost::bind( unbind, oldSolution ) );
        }
    }
}

/* ------------------------------------------------------------------------------------------------------------------ */
/*                                                  Jacobi halo                                                       */
/* ------------------------------------------------------------------------------------------------------------------ */

template<typename ValueType, bool useTexture>
__global__
void ell_jacobi_halo_kernel(
    ValueType* const solution,
    const ValueType* const diagonal,
    const int* const ellSizes,
    const int* const ellJA,
    const ValueType* const ellvalues,
    const int* const rowIndexes,
    const int numnonemptyrows,
    const int numrows,
    const ValueType* const oldsolution,
    const ValueType omega )
{
    const int id = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( id < numnonemptyrows )
    {
        int i = id;

        if ( rowIndexes )
        {
            i = rowIndexes[id];
        }

        ValueType temp = 0.0;

        int pos = i;
        const int rowend = ellSizes[i];

        for ( int jj = 0; jj < rowend; ++jj )
        {
            temp += ellvalues[pos] * fetchELLVectorX<ValueType, useTexture>( oldsolution, ellJA[pos] );
            pos += numrows;
        }

        const ValueType diag = diagonal[i];
        solution[i] -= temp * omega / diag;
    }
}

template<typename ValueType>
void CUDAELLUtils::jacobiHalo(
    ValueType solution[],
    const IndexType numRows,
    const ValueType diagonal[],
    const IndexType UNUSED( ellNumValuesPerRow ),
    const IndexType ellSizes[],
    const IndexType ellJA[],
    const ValueType ellValues[],
    const IndexType rowIndexes[],
    const IndexType numNonEmptyRows,
    const ValueType oldSolution[],
    const ValueType omega,
    SyncToken* syncToken )
{
    LAMA_REGION( "CUDA.ELL.jacobiHalo" )
 
    LAMA_LOG_INFO( logger, "jacobiHalo, #non-empty rows = " << numNonEmptyRows )

    LAMA_CHECK_CUDA_ACCESS

    const int blockSize = CUDASettings::getBlockSize( numNonEmptyRows );
    dim3 dimBlock( blockSize, 1, 1 );
    dim3 dimGrid = makeGrid( numNonEmptyRows, dimBlock.x );

    bool useTexture = CUDASettings::useTexture();

    if ( useTexture )
    {
        vectorELLBindTexture( oldSolution );

        LAMA_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( ell_jacobi_halo_kernel<ValueType), true>, hipFuncCachePreferL1 ),
                           "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )

        ell_jacobi_halo_kernel<ValueType, true> <<<dimGrid, dimBlock>>>(
            solution, diagonal, ellSizes, ellJA, ellValues,
            rowIndexes, numNonEmptyRows, numRows, oldSolution, omega );
    }
    else
    {
        LAMA_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( ell_jacobi_halo_kernel<ValueType), false>, hipFuncCachePreferL1 ),
                           "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" )

        ell_jacobi_halo_kernel<ValueType, false> <<<dimGrid, dimBlock>>>(
            solution, diagonal, ellSizes, ellJA, ellValues,
            rowIndexes, numNonEmptyRows, numRows, oldSolution, omega );
    }

    LAMA_CUDA_RT_CALL( hipGetLastError(), "LAMA_STATUS_ELLJACOBIHALO_CUDAKERNEL_FAILED" )
    LAMA_CUDA_RT_CALL( hipStreamSynchronize(0), "LAMA_STATUS_ELLJACOBIHALO_CUDAKERNEL_FAILED" )

    if ( useTexture )
    {
        vectorELLUnbindTexture( oldSolution );
    }
}

/* ------------------------------------------------------------------------------------------------------------------ */
/*                                Template instantiations via registration routine                                    */
/* ------------------------------------------------------------------------------------------------------------------ */

void CUDAELLUtils::setInterface( ELLUtilsInterface& ELLUtils )
{
    LAMA_LOG_INFO( logger, "set ELL routines for CUDA in Interface" )

    LAMA_INTERFACE_REGISTER( ELLUtils, countNonEmptyRowsBySizes )
    LAMA_INTERFACE_REGISTER( ELLUtils, setNonEmptyRowsBySizes )
    LAMA_INTERFACE_REGISTER( ELLUtils, hasDiagonalProperty )
    LAMA_INTERFACE_REGISTER( ELLUtils, check )

    LAMA_INTERFACE_REGISTER_TT( ELLUtils, getRow, float, float )
    LAMA_INTERFACE_REGISTER_TT( ELLUtils, getRow, float, double )
    LAMA_INTERFACE_REGISTER_TT( ELLUtils, getRow, double, float )
    LAMA_INTERFACE_REGISTER_TT( ELLUtils, getRow, double, double )

    LAMA_INTERFACE_REGISTER_TT( ELLUtils, getValue, float, float )
    LAMA_INTERFACE_REGISTER_TT( ELLUtils, getValue, float, double )
    LAMA_INTERFACE_REGISTER_TT( ELLUtils, getValue, double, float )
    LAMA_INTERFACE_REGISTER_TT( ELLUtils, getValue, double, double )

    LAMA_INTERFACE_REGISTER_TT( ELLUtils, scaleValue, float, float )
    LAMA_INTERFACE_REGISTER_TT( ELLUtils, scaleValue, double, float )
    LAMA_INTERFACE_REGISTER_TT( ELLUtils, scaleValue, float, double )
    LAMA_INTERFACE_REGISTER_TT( ELLUtils, scaleValue, double, double )

    LAMA_INTERFACE_REGISTER_TT( ELLUtils, setCSRValues, float, float )
    LAMA_INTERFACE_REGISTER_TT( ELLUtils, setCSRValues, double, float )
    LAMA_INTERFACE_REGISTER_TT( ELLUtils, setCSRValues, float, double )
    LAMA_INTERFACE_REGISTER_TT( ELLUtils, setCSRValues, double, double )

    LAMA_INTERFACE_REGISTER_TT( ELLUtils, getCSRValues, float, float )
    LAMA_INTERFACE_REGISTER_TT( ELLUtils, getCSRValues, float, double )
    LAMA_INTERFACE_REGISTER_TT( ELLUtils, getCSRValues, double, float )
    LAMA_INTERFACE_REGISTER_TT( ELLUtils, getCSRValues, double, double )

    LAMA_INTERFACE_REGISTER_T( ELLUtils, normalGEMV, float )
    LAMA_INTERFACE_REGISTER_T( ELLUtils, normalGEMV, double )

    LAMA_INTERFACE_REGISTER_T( ELLUtils, sparseGEMV, float )
    LAMA_INTERFACE_REGISTER_T( ELLUtils, sparseGEMV, double )

    LAMA_INTERFACE_REGISTER_T( ELLUtils, normalGEVM, float )
    LAMA_INTERFACE_REGISTER_T( ELLUtils, normalGEVM, double )

    LAMA_INTERFACE_REGISTER_T( ELLUtils, sparseGEVM, float )
    LAMA_INTERFACE_REGISTER_T( ELLUtils, sparseGEVM, double )

    LAMA_INTERFACE_REGISTER_T( ELLUtils, jacobi, float )
    LAMA_INTERFACE_REGISTER_T( ELLUtils, jacobi, double )

    LAMA_INTERFACE_REGISTER_T( ELLUtils, jacobiHalo, float )
    LAMA_INTERFACE_REGISTER_T( ELLUtils, jacobiHalo, double )

    LAMA_INTERFACE_REGISTER_T( ELLUtils, fillELLValues, float )
    LAMA_INTERFACE_REGISTER_T( ELLUtils, fillELLValues, double )
}

/* --------------------------------------------------------------------------- */
/*    Static registration of the Utils routines                                */
/* --------------------------------------------------------------------------- */

bool CUDAELLUtils::registerInterface()
{
    LAMAInterface& interface = LAMAInterfaceRegistry::getRegistry().modifyInterface( Context::CUDA );
    setInterface( interface.ELLUtils );
    return true;
}

/* --------------------------------------------------------------------------- */
/*    Static initialiazion at program start                                    */
/* --------------------------------------------------------------------------- */

bool CUDAELLUtils::initialized = registerInterface();

} // namespace lama
