
#include <hip/hip_runtime.h>
#include <omp.h>

__global__
void empty_kernel()
{
}

extern "C" double getKernelLaunchTime( int devNo )
{
    hipSetDevice( devNo );

    empty_kernel<<<1, 1>>>( );

    double time = omp_get_wtime();
    hipDeviceSynchronize();
    empty_kernel<<<1, 1>>>();
    hipDeviceSynchronize();
    return omp_get_wtime() - time;
}
