#include "hip/hip_runtime.h"
#include <lama/Walltime.hpp>

__global__
void empty_kernel()
{
}

extern "C" double getKernelLaunchTime( int devNo )
{
    hipSetDevice( devNo );

    empty_kernel<<<1, 1>>>( );

    double time = lama::Walltime::get();
    hipDeviceSynchronize();
    empty_kernel<<<1, 1>>>();
    hipDeviceSynchronize();
    return lama::Walltime::get() - time;
}
