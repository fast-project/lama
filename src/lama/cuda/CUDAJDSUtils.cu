#include "hip/hip_runtime.h"
/**
 * @file CUDAJDSUtils.cpp
 *
 * @license
 * Copyright (c) 2009-2013
 * Fraunhofer Institute for Algorithms and Scientific Computing SCAI
 * for Fraunhofer-Gesellschaft
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 * @endlicense
 *
 * @brief Implementation of JDS utilities with CUDA
 * @author Bea Hornef, Thomas Brandes
 * @date 04.07.2012
 * @since 1.0.0
 */

// hpp
#include <lama/cuda/utils.cu.h>

// others
#include <lama/cuda/CUDAStreamSyncToken.hpp>
#include <lama/cuda/CUDAError.hpp>
#include <lama/cuda/CUDASettings.hpp>
#include <lama/cuda/CUDAJDSUtils.hpp>
#include <lama/cuda/CUDAUtils.hpp>

#include <lama/exception/LAMAAssert.hpp>
#include <lama/tracing.hpp>

#include <lama/LAMAInterface.hpp>
#include <lama/LAMAInterfaceRegistry.hpp>

// macros
#include <lama/macros/unused.hpp>

// thrust
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/fill.h>
//#include <thrust/gather.h>
#include <thrust/iterator/reverse_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/reduce.h>
#include <thrust/scan.h>
#include <thrust/scatter.h>
#include <thrust/sort.h>
#include <thrust/transform_reduce.h>
#include <thrust/tuple.h>

namespace lama
{

LAMA_LOG_DEF_LOGGER( CUDAJDSUtils::logger, "CUDA.JDSUtils" )

/* ------------------------------------------------------------------------------------------------------------------ */
/*                                                  thrust functors                                                   */
/* ------------------------------------------------------------------------------------------------------------------ */

template<typename T>
struct identity
{
    const T x;
    identity( T _x )
        : x( _x )
    {
    }
    __host__ __device__
    T operator()( thrust::tuple<T,T> y )
    {
        if ( thrust::get < 0 > ( y ) == x )
        {
            return thrust::get < 1 > ( y );
        }
        return 0;
    }
};

template<typename T>
struct greaterThan
{
    const T x;
    greaterThan( T _x )
        : x( _x )
    {
    }

    __host__ __device__
    T operator()( T y )
    {
        return y > x;
    }
};

/* ------------------------------------------------------------------------------------------------------------------ */
/*                                                  getRow                                                            */
/* ------------------------------------------------------------------------------------------------------------------ */

template<typename ValueType,typename OtherValueType>
__global__
void getRowKernel(
    OtherValueType *row,
    const IndexType i,
    const IndexType *ilg,
    const IndexType *dlg,
    const IndexType *ja,
    const ValueType *values )
{
    IndexType offset = 0;

    for ( IndexType j = 0; j < ilg[i]; j++ )
    {
        row[ja[i + offset]] = static_cast<OtherValueType>( values[i + offset] );
        offset += dlg[j];
    }
}

template<typename ValueType,typename OtherValueType>
void CUDAJDSUtils::getRow(
    OtherValueType row[],
    const IndexType i,
    const IndexType numColumns,
    const IndexType numRows,
    const IndexType perm[],
    const IndexType ilg[],
    const IndexType dlg[],
    const IndexType ja[],
    const ValueType values[] )
{
    LAMA_LOG_INFO( logger, "getRow with i = " << i << ", numColumns = " << numColumns << " and numRows = " << numRows )

    LAMA_CHECK_CUDA_ACCESS

    thrust::device_ptr<OtherValueType> rowPtr( const_cast<OtherValueType*>( row ) );
    thrust::device_ptr<IndexType> permPtr( const_cast<IndexType*>( perm ) );

    thrust::fill( rowPtr, rowPtr + numColumns, static_cast<OtherValueType>( 0 ) );

    thrust::counting_iterator<IndexType> sequence( 0 );

    // correct index with permutation array
    IndexType ii = thrust::transform_reduce(
                       thrust::make_zip_iterator( thrust::make_tuple( permPtr, sequence ) ),
                       thrust::make_zip_iterator( thrust::make_tuple( permPtr + numRows, sequence + numRows ) ),
                       identity<IndexType>( i ), 0, thrust::plus<IndexType>() );

    const int block_size = 256;
    dim3 dimBlock( block_size, 1, 1 );
    dim3 dimGrid = makeGrid( 1, dimBlock.x );

    //TODO: find better CUDA / Thrust implementation
    getRowKernel<<<dimGrid, dimBlock>>>( row, ii, ilg, dlg, ja, values );

    LAMA_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "JDS:getRowKernel FAILED" )
}

/* ------------------------------------------------------------------------------------------------------------------ */
/*                                                  getValue                                                          */
/* ------------------------------------------------------------------------------------------------------------------ */

template<typename ValueType>
__global__
void getValueKernel(
    const IndexType i,
    const IndexType j,
    const IndexType numRows,
    const IndexType* dlg,
    const IndexType* ilg,
    const IndexType* perm,
    const IndexType* ja,
    const ValueType* values,
    ValueType* result )
{
    const int tId = threadId( gridDim, blockIdx, blockDim, threadIdx );
    if ( tId == 0 )
    {
        IndexType ii;

        // check the permutation of row i
        for ( ii = 0; ii < numRows; ii++ )
        {
            if ( perm[ii] == i )
            {
                break;
            }
        }

        IndexType k = 0;
        bool found = false;

        for ( IndexType jj = 0; jj < ilg[ii]; jj++ )
        {
            if ( ja[ii + k] == j )
            {
                result[0] = values[ii + k];
                found = true;
                break;
            }

            k += dlg[jj];
        }

        if ( !found )
        {
            result[0] = 0.0;
        }
    }
}

template<typename ValueType,typename NoType>
ValueType CUDAJDSUtils::getValue(
    const IndexType i,
    const IndexType j,
    const IndexType numRows,
    const IndexType* dlg,
    const IndexType* ilg,
    const IndexType* perm,
    const IndexType* ja,
    const ValueType* values )
{
    LAMA_CHECK_CUDA_ACCESS

    thrust::device_ptr<ValueType> resultPtr = thrust::device_malloc < ValueType > ( 1 );
    ValueType *resultRawPtr = thrust::raw_pointer_cast( resultPtr );

    const int block_size = 256;
    dim3 dimBlock( block_size, 1, 1 );
    dim3 dimGrid = makeGrid( 1, dimBlock.x );

    //TODO: find better CUDA / Thrust implementation
    getValueKernel<<<dimGrid, dimBlock>>>( i, j, numRows, dlg, ilg, perm, ja, values, resultRawPtr );

    thrust::host_vector<ValueType> resultHost( resultPtr, resultPtr + 1 );

    return resultHost[0];
}

/* ------------------------------------------------------------------------------------------------------------------ */
/*                                                  scaleValue                                                        */
/* ------------------------------------------------------------------------------------------------------------------ */

template<typename ValueType,typename OtherValueType>
__global__
void scaleValueKernel(
    const IndexType numRows,
    const IndexType *perm,
    const IndexType *ilg,
    const IndexType *dlg,
    ValueType *mValues,
    const OtherValueType *values )
{
    const int i = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( i < numRows )
    {
        IndexType offset = i;
        OtherValueType value = values[perm[i]];

        for ( IndexType j = 0; j < ilg[i]; j++ )
        {
            mValues[offset] *= static_cast<ValueType>( value );
            offset += dlg[j];
        }
    }
}

template<typename ValueType,typename OtherValueType>
void CUDAJDSUtils::scaleValue(
    const IndexType numRows,
    const IndexType perm[],
    const IndexType ilg[],
    const IndexType dlg[],
    ValueType mValues[],
    const OtherValueType values[] )
{
    LAMA_LOG_INFO( logger, "scaleValue with numRows = " << numRows )

    LAMA_CHECK_CUDA_ACCESS

    const int block_size = 256;
    dim3 dimBlock( block_size, 1, 1 );
    dim3 dimGrid = makeGrid( numRows, dimBlock.x );

    scaleValueKernel<<<dimGrid, dimBlock>>>( numRows, perm, ilg, dlg, mValues, values );

    LAMA_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "JDS:scaleValueKernel FAILED" )
}

/* ------------------------------------------------------------------------------------------------------------------ */
/*                                                  checkDiagonalProperty                                             */
/* ------------------------------------------------------------------------------------------------------------------ */

__global__
void checkDiagonalPropertyKernel( 
    bool *result, 
    const IndexType numRows, 
    const IndexType numColumns,
    const IndexType nonEmptyRows,
    const IndexType *perm, 
    const IndexType *ja )
{
    const int i = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( i >= numRows )
    {
        return;
    }

    const IndexType iRow = perm[i];

    if ( iRow >= numColumns )
    {
        // row does not count for diagonal

        return;
    }

    if ( i >= nonEmptyRows )
    {
        // iRow has no entries at all, ilg[i] is 0

        result[0] = false;
    }
    else if ( ja[i] != iRow )
    {
        result[0] = false;
    }
}

bool CUDAJDSUtils::checkDiagonalProperty(
    const IndexType numDiagonals,
    const IndexType numRows,
    const IndexType numColumns,
    const IndexType perm[],
    const IndexType ja[],
    const IndexType dlg[] )
{
    LAMA_LOG_INFO( logger, "checkDiagonalProperty with numDiagonals = " << numDiagonals 
                     << ", numRows = " << numRows << " and numColumns = " << numColumns )

    LAMA_CHECK_CUDA_ACCESS

    if ( numRows <= 0 ) 
    {
        return false;
    }

    if ( numDiagonals <= 0 )
    {
        return false;
    }

    // now it is sure that dlg, perm and ja are not empty

    const IndexType diagSize = std::min( numRows, numColumns );

    IndexType nonEmptyRows = 0;

    LAMA_CUDA_RT_CALL( hipMemcpy( &nonEmptyRows, &dlg[0], sizeof( IndexType ), hipMemcpyDeviceToHost ),
                       "get number of non-zero rows from dlg" );

    // Be careful: numDiagonals has nothing to do with size of diagonal

    if ( nonEmptyRows < diagSize )
    {
         return false;
    }

    bool* d_hasProperty;   // will be ptr to device version of hasProperty

    bool hasProperty = true;

    LAMA_CUDA_RT_CALL( hipMalloc( (void**) &d_hasProperty, sizeof( bool ) ),
                       "allocate 4 bytes on the device for the result of hasDiagonalProperty_kernel" )

    LAMA_CUDA_RT_CALL( hipMemcpy( d_hasProperty, &hasProperty, sizeof( bool ), hipMemcpyHostToDevice ),
                       "copy flag for diagonalProperty to device" )

    const int block_size = 256;
    dim3 dimBlock( block_size, 1, 1 );
    dim3 dimGrid = makeGrid( numRows, dimBlock.x );

    checkDiagonalPropertyKernel<<<dimGrid, dimBlock>>>( d_hasProperty,
                                                        numRows, numColumns, nonEmptyRows,
                                                        perm, ja );

    LAMA_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "JDS:checkDiagonalPropertyKernel FAILED" )

    LAMA_CUDA_RT_CALL( hipMemcpy( &hasProperty, d_hasProperty, sizeof( bool ), hipMemcpyDeviceToHost ),
                       "copy flag for diagonalProperty to host" )

    LAMA_CUDA_RT_CALL( hipFree( d_hasProperty ),
                       "free result var for diagonal property" )

    return hasProperty;
}

/* ------------------------------------------------------------------------------------------------------------------ */
/*                                                  ilg2dlg                                                           */
/* ------------------------------------------------------------------------------------------------------------------ */

__global__
void ilg2dlgKernel( IndexType *dlg, const IndexType numDiagonals, const IndexType *ilg, const IndexType numRows )
{
    // Entries in dlg filled every time there is a change in values of consecutive elements
    //   i:     0  1  2  3  4  5
    // ilg:     5  5  3  3  3  1
    // nd1:     5  5  3  3  3  1
    // nd2:     5  3  3  3  1  0
    //             x        x  x
    //             |        |  |->    6 
    //             |        |---->       5  5 
    //             |------------->             2   2
    // dlg:                           6  5  5  2   2

    const int i = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( i < numRows )
    {
        IndexType nd1 = ilg[i];
        IndexType nd2 = 0;
 
        if ( i + 1 < numRows )
        {
            nd2 = ilg[i + 1];
        }

        for ( IndexType j = nd2; j < nd1; j++ )
        {
            dlg[j] = i + 1;
        }
    }
}

IndexType CUDAJDSUtils::ilg2dlg(
    IndexType dlg[],
    const IndexType numDiagonals,
    const IndexType ilg[],
    const IndexType numRows )
{
    LAMA_REGION( "CUDA.JDS:dlg<-ilg" )

    LAMA_LOG_INFO( logger, "ilg2dlg with numDiagonals = " << numDiagonals << ", numRows = " << numRows )

    LAMA_CHECK_CUDA_ACCESS

    if ( numDiagonals == 0 )
    {
        return 0;
    }

    // wrap raw pointer ilg to build sum, const_cast required, is safe 

    thrust::device_ptr<IndexType> ilgPtr( const_cast<IndexType*>( ilg ) );

    IndexType sumIlg = thrust::reduce( ilgPtr, ilgPtr + numRows, 0, thrust::plus<IndexType>() );

    const int block_size = 256;
    dim3 dimBlock( block_size, 1, 1 );
    dim3 dimGrid = makeGrid( numRows, dimBlock.x );

    ilg2dlgKernel<<<dimGrid, dimBlock>>>( dlg, numDiagonals, ilg, numRows );

    LAMA_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "JDS: ilg2dlgKernel FAILED" )

    return sumIlg;
}

/* ------------------------------------------------------------------------------------------------------------------ */
/*                                                  sortRows                                                          */
/* ------------------------------------------------------------------------------------------------------------------ */

void CUDAJDSUtils::sortRows( IndexType array[], IndexType perm[], const IndexType n )
{
    LAMA_REGION( "CUDA.JDS:sortRows" )

    LAMA_LOG_INFO( logger, "sort " << n << " rows by sizes" )

    LAMA_CHECK_CUDA_ACCESS
    thrust::device_ptr<IndexType> array_d( const_cast<IndexType*>( array ) );
    thrust::device_ptr<IndexType> perm_d( const_cast<IndexType*>( perm ) );

    thrust::stable_sort_by_key( array_d, array_d + n, perm_d, thrust::greater<IndexType>() );

    LAMA_CUDA_RT_CALL( hipStreamSynchronize(0), "JDS: ilg2dlgKernel FAILED" )
}

/* ------------------------------------------------------------------------------------------------------------------ */
/*                                                  setCSRValues                                                      */
/* ------------------------------------------------------------------------------------------------------------------ */

template<typename JDSValueType, typename CSRValueType, bool useSharedMem>
__global__
void csr2jdsKernel(
    IndexType* jdsJa,
    JDSValueType* jdsValues,
    const IndexType* const jdsDLG,
    const IndexType  ndlg,
    const IndexType* const jdsILG,
    const IndexType* const jdsPerm,
    const IndexType nrows,
    const IndexType* const csrIa,
    const IndexType* const csrJa,
    const CSRValueType* const csrValues )
{
    extern __shared__ int dlg[];

    const IndexType iJDS = threadId( gridDim, blockIdx, blockDim, threadIdx );

    // copy DLG array into shared memory for faster access

    if ( useSharedMem )
    {
        int k = threadIdx.x;
        while ( k < ndlg )
        {
            dlg[k] = jdsDLG[k];
            k += blockDim.x;
        }
        __syncthreads();
    }

    if ( iJDS < nrows )
    {
        const IndexType iCSR = jdsPerm[iJDS];  // row index for CSR data

        const IndexType csrOffset = csrIa[iCSR];

        IndexType jdsOffset = iJDS;

        const IndexType numValuesInRow = jdsILG[iJDS];

        for ( IndexType jj = 0; jj < numValuesInRow; ++jj )
        {
            jdsJa[jdsOffset] = csrJa[csrOffset + jj];
            jdsValues[jdsOffset] = static_cast<JDSValueType>( csrValues[csrOffset + jj] );

            if ( useSharedMem ) 
            {
                jdsOffset += dlg[jj];
            }
            else
            {
                jdsOffset += jdsDLG[jj];
            }
        }
    }
}

template<typename JDSValueType, typename CSRValueType>
void CUDAJDSUtils::setCSRValues(
    IndexType jdsJA[],
    JDSValueType jdsValues[],
    const IndexType numRows,
    const IndexType jdsPerm[],
    const IndexType jdsILG[],
    const IndexType ndlg,
    const IndexType jdsDLG[],
    const IndexType csrIA[],
    const IndexType csrJA[],
    const CSRValueType csrValues[] )
{
    // convert CSR data to JDS, ja and values

    LAMA_REGION( "CUDA.JDS<-CSR_values" )

    LAMA_LOG_INFO( logger, "convert CSR to JDS, #rows = " << numRows )

    LAMA_CHECK_CUDA_ACCESS

    bool useSharedMem = CUDASettings::useSharedMem();

    const int block_size = 256;
    dim3 dimBlock( block_size, 1, 1 );
    dim3 dimGrid = makeGrid( numRows, dimBlock.x );

    LAMA_LOG_INFO( logger, "Start csr2jds_kernel<" << typeid( JDSValueType ).name()
                           << ", " << typeid( CSRValueType ).name()
                           << ", useSharedMem = " << useSharedMem
                           << "> ( nrows = " << numRows << ", ndiag = " << ndlg << " )" );

    if ( useSharedMem )
    {
        LAMA_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( csr2jdsKernel<JDSValueType), CSRValueType, true>,
                                                   hipFuncCachePreferL1 ),
                           "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" );

        const int sharedMemSize = ndlg * sizeof(int);

        csr2jdsKernel<JDSValueType, CSRValueType, true><<<dimGrid, dimBlock, sharedMemSize>>>(
            jdsJA, jdsValues, jdsDLG, ndlg, jdsILG, jdsPerm, numRows, csrIA, csrJA, csrValues );
    }
    else
    {
        LAMA_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( csr2jdsKernel<JDSValueType), CSRValueType, false>,
                                                   hipFuncCachePreferL1 ),
                           "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" );

        csr2jdsKernel<JDSValueType, CSRValueType, false><<<dimGrid, dimBlock, 0>>>(
            jdsJA, jdsValues, jdsDLG, ndlg, jdsILG, jdsPerm, numRows, csrIA, csrJA, csrValues );
    }

    LAMA_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "csr2jdsKernel failed" );

    LAMA_LOG_INFO( logger, "Ready csr2jds_kernel<" << typeid( JDSValueType ).name() 
                           << ", " << typeid( CSRValueType ).name() <<  " )" )
}

/* ------------------------------------------------------------------------------------------------------------------ */
/*                                                  setInversePerm                                                    */
/* ------------------------------------------------------------------------------------------------------------------ */

void CUDAJDSUtils::setInversePerm( IndexType inversePerm[], const IndexType perm[], const IndexType n )
{
    LAMA_LOG_INFO( logger, "compute inverse perm, n = " << n )

    LAMA_CHECK_CUDA_ACCESS

    if ( n > 0 )
    {
        thrust::device_ptr<IndexType> inversePermPtr( const_cast<IndexType*>( inversePerm ) );
        thrust::device_ptr<IndexType> permPtr( const_cast<IndexType*>( perm ) );

        thrust::counting_iterator<IndexType> sequence( 0 );

        thrust::scatter( sequence, sequence + n, permPtr, inversePermPtr );

        LAMA_CHECK_CUDA_ERROR
    }
}

/* ------------------------------------------------------------------------------------------------------------------ */
/*                                                  getCSRValues                                                      */
/* ------------------------------------------------------------------------------------------------------------------ */

template<typename JDSValueType,typename CSRValueType>
__global__
void jds2csrKernel(
    IndexType *csrJA,
    CSRValueType *csrValues,
    const IndexType *csrIA,
    const IndexType numRows,
    const IndexType *jdsInversePerm,
    const IndexType *jdsILG,
    const IndexType *jdsDLG,
    const IndexType *jdsJA,
    const JDSValueType *jdsValues )
{
    const int i = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( i < numRows )
    {
        IndexType ii = jdsInversePerm[i]; // where to find row i in JDS storage

        const IndexType numValuesInRow = jdsILG[ii];

        IndexType jdsOffset = ii; // run through input JDS data
        IndexType offset = csrIA[i]; // run through output data

        for ( IndexType jj = 0; jj < numValuesInRow; jj++ )
        {
            csrJA[offset + jj] = jdsJA[jdsOffset];
            csrValues[offset + jj] = static_cast<CSRValueType>( jdsValues[jdsOffset] );
            jdsOffset += jdsDLG[jj];
        }

    }
}

template<typename JDSValueType,typename CSRValueType>
void CUDAJDSUtils::getCSRValues(
    IndexType csrJA[],
    CSRValueType csrValues[],
    const IndexType csrIA[],
    const IndexType numRows,
    const IndexType jdsInversePerm[],
    const IndexType jdsILG[],
    const IndexType jdsDLG[],
    const IndexType jdsJA[],
    const JDSValueType jdsValues[] )
{
    LAMA_REGION( "CUDA.JDS->CSR_values" )

    LAMA_LOG_INFO( logger,
                   "get CSRValues<" << typeid( JDSValueType ).name() << ", " << typeid( CSRValueType ).name() << ">" << ", #rows = " << numRows )

    LAMA_CHECK_CUDA_ACCESS

    const int block_size = 256;
    dim3 dimBlock( block_size, 1, 1 );
    dim3 dimGrid = makeGrid( numRows, dimBlock.x );

    jds2csrKernel<<<dimGrid,dimBlock>>>( csrJA, csrValues, csrIA, numRows, jdsInversePerm, jdsILG, jdsDLG, jdsJA,
                                         jdsValues );

    LAMA_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "JDS:jds2csrKernel FAILED" )
}

/* ------------------------------------------------------------------------------------------------------------------ */
/*                                                  xxxxx                                                             */
/* ------------------------------------------------------------------------------------------------------------------ */

/* --------------------------------------------------------------------------- */
/*                          Jacobi                                             */
/* --------------------------------------------------------------------------- */

texture<float,1> textureJDSFloatXRef;

texture<int2,1> textureJDSDoubleXRef;

texture<int,1> texJDSdlgRef;

__inline__ void jdsBindTexture( const float* vector )
{
    LAMA_CUDA_RT_CALL( hipBindTexture( NULL, textureJDSFloatXRef, vector ), "LAMA_STATUS_CUDA_BINDTEX_FAILED" )
}

__inline__ void jdsBindTexture( const double* vector )
{
    LAMA_CUDA_RT_CALL( hipBindTexture( NULL, textureJDSDoubleXRef, vector ), "LAMA_STATUS_CUDA_BINDTEX_FAILED" )
}


__inline__ void jdsUnbindTexture( const float* )
{
    LAMA_CUDA_RT_CALL( hipUnbindTexture( textureJDSFloatXRef ), "LAMA_STATUS_CUDA_UNBINDTEX_FAILED" )
}

__inline__ void jdsUnbindTexture( const double* )
{
    LAMA_CUDA_RT_CALL( hipUnbindTexture( textureJDSDoubleXRef ), "LAMA_STATUS_CUDA_UNBINDTEX_FAILED" )
}

/* --------------------------------------------------------------------------- */

template<typename T,bool useTexture>
__inline__    __device__ T fetch_JDSx( const T* const x, const int i )
{
    return x[i];
}

template<bool useTexture,bool useSharedMemory>
__inline__ __device__
int fetch_JDSdlg( const int* const dlg_d, int[], const int i )
{
    return dlg_d[i];
}

template<>
__inline__ __device__
float fetch_JDSx<float,true>( const float* const, const int i )
{
    return tex1Dfetch( textureJDSFloatXRef, i );
}

template<>
__inline__ __device__
double fetch_JDSx<double,true>( const double* const, const int i )
{
    int2 v = tex1Dfetch( textureJDSDoubleXRef, i );
    return __hiloint2double( v.y, v.x );
}

template<>
__inline__ __device__
int fetch_JDSdlg<true,false>( const int* const, int[], const int i )
{
    return tex1Dfetch( texJDSdlgRef, i );
}

template<>
__inline__ __device__
int fetch_JDSdlg<true,true>( const int* const, int dlg_sm[], const int i )
{
    return dlg_sm[i];
}

template<>
__inline__ __device__
int fetch_JDSdlg<false,true>( const int* const, int dlg_sm[], const int i )
{
    return dlg_sm[i];
}

template<typename T,bool useTexture,bool useSharedMem>
__global__
void jds_jacobi_kernel(
    const T* const jdsValues,
    const int* const jdsDLG,
    const int ndlg,
    const int* const jdsIlg,
    const int* const jdsJA,
    const int* const jdsPerm,
    const int numRows,
    const T* const rhs,
    T* const solution,
    const T* const oldSolution,
    const T omega )
{
    extern __shared__ int dlg[];
    const int i = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( useSharedMem )
    {
        int k = threadIdx.x;
        while ( k < ndlg )
        {
            dlg[k] = jdsDLG[k];
            k += blockDim.x;
        }
        __syncthreads();
    }

    if ( i < numRows )
    {
        const int perm = jdsPerm[i];

        T temp = rhs[perm];

        const T aDiag = jdsValues[i];

        int pos = i + fetch_JDSdlg<useTexture,useSharedMem>( jdsDLG, dlg, 0 );
        const int rowEnd = jdsIlg[i];
        for ( int jj = 1; jj < rowEnd; ++jj )
        {
            temp -= jdsValues[pos] * fetch_JDSx<T,useTexture>( oldSolution, jdsJA[pos] );
            pos += fetch_JDSdlg<useTexture,useSharedMem>( jdsDLG, dlg, jj );
        }

        if ( omega == 0.5 )
        {
            solution[perm] = omega * ( fetch_JDSx<T,useTexture>( oldSolution, perm ) + temp / aDiag );
        }
        else if ( omega == 1.0 )
        {
            solution[perm] = temp / aDiag;
        }
        else
        {
            solution[perm] = omega * ( temp / aDiag ) + ( 1.0 - omega ) * fetch_JDSx<T,useTexture>( oldSolution, perm );
        }

    }
}

template<typename ValueType>
void CUDAJDSUtils::jacobi(
    ValueType solution[],
    const IndexType numRows,
    const IndexType jdsPerm[],
    const IndexType jdsIlg[],
    const IndexType ndlg,
    const IndexType jdsDLG[],
    const IndexType jdsJA[],
    const ValueType jdsValues[],
    const ValueType oldSolution[],
    const ValueType rhs[],
    const ValueType omega,
    SyncToken* syncToken )
{
    LAMA_REGION( "CUDA.JDS.jacobi" )

    hipStream_t stream = 0;

    LAMA_LOG_INFO( logger,
                   "jacobi<" << typeid(ValueType).name() << ">" << ", #rows = " << numRows << ", omega = " << omega )

    LAMA_CHECK_CUDA_ACCESS

    if ( syncToken )
    {
        CUDAStreamSyncToken* cudaStreamSyncToken = dynamic_cast<CUDAStreamSyncToken*>( syncToken );
        LAMA_ASSERT_DEBUG( cudaStreamSyncToken, "no cuda stream sync token provided" )
        stream = cudaStreamSyncToken->getCUDAStream();
    }

    const int block_size = ( numRows > 8191 ? 256 : 128 );
    dim3 dimBlock( block_size, 1, 1 );
    dim3 dimGrid = makeGrid( numRows, dimBlock.x );

    bool useTexture = CUDASettings::useTexture();

    if ( syncToken )
    {
        // asycnronous operation not supported with textures ( free must be done dynamically )

        useTexture = false;
    }

    const bool useSharedMem = CUDASettings::useSharedMem();

    LAMA_LOG_DEBUG( logger, "useTexture = " << useTexture << ", useSharedMem = " << useSharedMem )

    if ( useTexture )
    {
        jdsBindTexture( oldSolution );

        if ( !useSharedMem )
        {
            LAMA_CUDA_RT_CALL( hipBindTexture( NULL, texJDSdlgRef, jdsDLG ), "LAMA_STATUS_CUDA_BINDTEX_FAILED" );
            LAMA_CUDA_RT_CALL(
                hipFuncSetCacheConfig(reinterpret_cast<const void*>( jds_jacobi_kernel<ValueType), true, false>, hipFuncCachePreferL1 ),
                "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" );
        }
        else
        {
            LAMA_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( jds_jacobi_kernel<ValueType), true, true>,hipFuncCachePreferL1 ),
                               "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" );
        }
    }
    else
    {
        if ( !useSharedMem )
        {
            LAMA_CUDA_RT_CALL(
                hipFuncSetCacheConfig(reinterpret_cast<const void*>( jds_jacobi_kernel<ValueType), false, false>,hipFuncCachePreferL1 ),
                "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" );
        }
        else
        {
            LAMA_CUDA_RT_CALL(
                hipFuncSetCacheConfig(reinterpret_cast<const void*>( jds_jacobi_kernel<ValueType), false, true>, hipFuncCachePreferL1 ),
                "LAMA_STATUS_CUDA_FUNCSETCACHECONFIG_FAILED" );
        }
    }

    LAMA_LOG_INFO( logger, "Start jds_jacobi_kernel<" << typeid( ValueType ).name() 
                           << ", useTexture = " << useTexture << ", useSharedMem = " << useSharedMem << ">" );

    if ( useTexture )
    {
        if ( !useSharedMem )
        {
            jds_jacobi_kernel<ValueType, true, false> <<<dimGrid, dimBlock, 0, stream>>>( 
                jdsValues, jdsDLG, ndlg, jdsIlg, jdsJA, jdsPerm, numRows, rhs, solution, oldSolution, omega );
        }
        else
        {
            const int sharedMemSize = ndlg * sizeof(int);
            jds_jacobi_kernel<ValueType, true, true> <<<dimGrid, dimBlock, sharedMemSize, stream>>>( 
                jdsValues, jdsDLG, ndlg, jdsIlg, jdsJA, jdsPerm, numRows, rhs, solution, oldSolution, omega );
        }
    }
    else
    {
        if ( !useSharedMem )
        {
            jds_jacobi_kernel<ValueType, false, false> <<<dimGrid, dimBlock, 0, stream>>>( 
                jdsValues, jdsDLG, ndlg, jdsIlg, jdsJA, jdsPerm, numRows, rhs, solution, oldSolution, omega );
        }
        else
        {
            const int sharedMemSize = ndlg * sizeof(int);
            jds_jacobi_kernel<ValueType, false, true> <<<dimGrid, dimBlock, sharedMemSize, stream>>>( 
                jdsValues, jdsDLG, ndlg, jdsIlg, jdsJA, jdsPerm, numRows, rhs, solution, oldSolution, omega);
        }
    }

    LAMA_CUDA_RT_CALL( hipGetLastError(), "jds_jacobi_kernel<" << typeid( ValueType ).name() 
                                           << ", " << useTexture << ", " << useSharedMem << "> failed" )

    if ( !syncToken )
    {
        LAMA_CUDA_RT_CALL( hipStreamSynchronize( stream ), "JDS:jacobi_kernel failed" )
    }

    if ( useTexture )
    {
        jdsUnbindTexture( oldSolution );

        if ( !useSharedMem )
        {
            LAMA_CUDA_RT_CALL( hipUnbindTexture( texJDSdlgRef ), "LAMA_STATUS_CUDA_UNBINDTEX_FAILED" )
        }
    }
}

/* --------------------------------------------------------------------------- */
/*                          Jacobi halo                                        */
/* --------------------------------------------------------------------------- */

template<typename T,bool useTexture,bool useSharedMem>
__global__
void jds_jacobi_halo_kernel(
    const T* const diagonal,
    const T* const jdsValuesHalo,
    const int* const jdsDLGHalo,
    const int ndlg_halo,
    const int* const jdsIlgHalo,
    const int* const jdsJAHalo,
    const int* const jdsPermHalo,
    T* const solutionLocal,
    const T* const oldSolutionHalo,
    const T omega )
{
    extern __shared__ int dlg[];

    const int id = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( useSharedMem )
    {
        int k = threadIdx.x;
        while ( k < ndlg_halo )
        {
            dlg[k] = jdsDLGHalo[k];
            k += blockDim.x;
        }
        __syncthreads();
    }

    if ( id < fetch_JDSdlg<useTexture,useSharedMem>( jdsDLGHalo, dlg, 0 ) )
    {
        T temp = 0.0;
        int pos = id;
        const int rowEnd = jdsIlgHalo[id];
        const int perm = jdsPermHalo[id];
        for ( int jj = 0; jj < rowEnd; ++jj )
        {
            temp += jdsValuesHalo[pos] * fetch_JDSx<T,useTexture>( oldSolutionHalo, jdsJAHalo[pos] );
            pos += fetch_JDSdlg<useTexture,useSharedMem>( jdsDLGHalo, dlg, jj );
        }

        const T aDiag = diagonal[perm];
        solutionLocal[perm] -= temp * omega / aDiag;
    }
}

template<typename ValueType>
void CUDAJDSUtils::jacobiHalo(
    ValueType solutionLocal[],
    const IndexType numRows,
    const ValueType diagonal[],
    const IndexType ndlg_halo,
    const IndexType jdsPermHalo[],
    const IndexType jdsIlgHalo[],
    const IndexType jdsDLGHalo[],
    const IndexType jdsJAHalo[],
    const ValueType jdsValuesHalo[],
    const ValueType oldSolutionHalo[],
    const ValueType omega,
    SyncToken* UNUSED(syncToken) )
{
    LAMA_REGION( "CUDA.JDS.jacobiHalo" )

    LAMA_LOG_INFO( logger, "jacobiHalo<" << typeid(ValueType).name() << ">" 
                            << ", #rows = " << numRows << ", omega = " << omega )

    LAMA_CHECK_CUDA_ACCESS

    const int block_size = ( numRows > 8191 ? 256 : 128 ) / 2;
    dim3 dimBlock( block_size, 1, 1 );
    dim3 dimGrid = makeGrid( numRows, dimBlock.x ); // TODO:numRows is too much...

    bool useTexture   = CUDASettings::useTexture();
    bool useSharedMem = CUDASettings::useSharedMem(); 

    LAMA_LOG_DEBUG( logger, "useTexture = " << useTexture << ", useSharedMem = " << useSharedMem )

    if ( useTexture )
    {
        jdsBindTexture( oldSolutionHalo );

        if ( !useSharedMem )
        {
            LAMA_CUDA_RT_CALL( hipBindTexture( NULL, texJDSdlgRef, jdsDLGHalo ), "LAMA_STATUS_CUDA_BINDTEX_FAILED" );
        }
    }

    LAMA_LOG_INFO( logger, "Start jds_jacobi_halo_kernel<" << typeid( ValueType ).name() 
                           << ", useTexture = " << useTexture << ", useSharedMem = " << useSharedMem << ">" );

    if ( useTexture )
    {
        if ( !useSharedMem )
        {
            LAMA_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( jds_jacobi_halo_kernel<ValueType), true, false>, 
                                                       hipFuncCachePreferL1),
                               "hipFuncSetCacheConfig jds_jacobi_halo_kernel<ValueType, true, false> failed" )

            jds_jacobi_halo_kernel<ValueType, true, false> <<<dimGrid, dimBlock, 0>>>( 
                diagonal, jdsValuesHalo, jdsDLGHalo, ndlg_halo, jdsIlgHalo, jdsJAHalo,
                jdsPermHalo, solutionLocal, oldSolutionHalo, omega);
        }
        else
        {
            LAMA_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( jds_jacobi_halo_kernel<ValueType), true, true>, 
                                                       hipFuncCachePreferL1),
                               "hipFuncSetCacheConfig jds_jacobi_halo_kernel<ValueType, true, true> failed" )

            const int sharedMemSize = ndlg_halo * sizeof(int);

            jds_jacobi_halo_kernel<ValueType, true, true> <<<dimGrid, dimBlock, sharedMemSize>>>( 
                diagonal, jdsValuesHalo, jdsDLGHalo, ndlg_halo, jdsIlgHalo, jdsJAHalo,
                jdsPermHalo, solutionLocal, oldSolutionHalo, omega);
        }

    }
    else
    {
        if ( !useSharedMem )
        {
            LAMA_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( jds_jacobi_halo_kernel<ValueType), false, false>, 
                                                       hipFuncCachePreferL1),
                               "hipFuncSetCacheConfig jds_jacobi_halo_kernel<ValueType, false, false> failed" )

            jds_jacobi_halo_kernel<ValueType, false, false> <<<dimGrid,dimBlock>>>( 
                diagonal, jdsValuesHalo, jdsDLGHalo, ndlg_halo, jdsIlgHalo, jdsJAHalo,
                jdsPermHalo, solutionLocal, oldSolutionHalo, omega);
        }
        else
        {
            LAMA_CUDA_RT_CALL( hipFuncSetCacheConfig(reinterpret_cast<const void*>( jds_jacobi_halo_kernel<ValueType), false, true>, 
                                                       hipFuncCachePreferL1),
                               "hipFuncSetCacheConfig jds_jacobi_halo_kernel<ValueType, false, true> failed" )

            const int sharedMemSize = ndlg_halo * sizeof(int);

            jds_jacobi_halo_kernel<ValueType, false, true> <<<dimGrid, dimBlock, sharedMemSize>>>(
                diagonal, jdsValuesHalo, jdsDLGHalo, ndlg_halo, jdsIlgHalo, jdsJAHalo,
                jdsPermHalo, solutionLocal, oldSolutionHalo, omega);
        }
    }

    LAMA_CUDA_RT_CALL( hipStreamSynchronize(0), "jds_jacobi_halo_kernel" );

    if ( useTexture )
    {
        jdsUnbindTexture( oldSolutionHalo );

        if ( !useSharedMem )
        {
            LAMA_CUDA_RT_CALL( hipUnbindTexture( texJDSdlgRef ), "LAMA_STATUS_CUDA_UNBINDTEX_FAILED" );
        }
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType,bool useTexture,bool useSharedMem>
__global__
void jdsgemvKernel(
    IndexType n,
    const ValueType alpha,
    const ValueType* const jdsValues,
    const IndexType* const jdsDLG,
    const IndexType ndlg,
    const IndexType* const jdsIlg,
    const IndexType* jdsJA,
    const IndexType* jdsPerm,
    const ValueType* x_d,
    const ValueType beta,
    const ValueType* y_d,
    ValueType* const result_d )
{
    extern __shared__ IndexType dlg[];
    const IndexType ii = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( useSharedMem )
    {
        int k = threadIdx.x;
        while ( k < ndlg )
        {
            dlg[k] = jdsDLG[k];
            k += blockDim.x;
        }
        __syncthreads();
    }

    if ( ii < n )
    {
        IndexType i = jdsPerm[ii];  // row in matrix

        ValueType summand = 0.0;

        if ( beta != 0.0 )
        {
            summand = beta * y_d[i];
        }

        ValueType value = 0.0;

        int pos = ii;  // position in jdsJA, jdsValues

        int ni  = jdsIlg[ii];  // number entries in row

        for ( int jj = 0; jj < ni; ++jj )
        {
            IndexType j = jdsJA[pos];
            value += jdsValues[pos] * fetch_JDSx<ValueType,useTexture>( x_d, j );
            pos += fetch_JDSdlg<useTexture,useSharedMem>( jdsDLG, dlg, jj );
        }

        result_d[i] = alpha * value + summand;
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType,bool useTexture,bool useSharedMem>
__global__
void jdsgemvSparseKernel(
    IndexType numNonZeroRows,
    const ValueType alpha,
    const ValueType* const jdsValues,
    const IndexType* const jdsDLG,
    const IndexType ndlg,
    const IndexType* const jdsIlg,
    const IndexType* jdsJA,
    const IndexType* jdsPerm,
    const ValueType* x_d,
    ValueType* const result_d )
{
    extern __shared__ IndexType dlg[];
    const IndexType ii = threadId( gridDim, blockIdx, blockDim, threadIdx );

    if ( useSharedMem )
    {
        int k = threadIdx.x;
        while ( k < ndlg )
        {
            dlg[k] = jdsDLG[k];
            k += blockDim.x;
        }
        __syncthreads();
    }

    if ( ii < numNonZeroRows )
    {
        IndexType i = jdsPerm[ii];  // row in matrix

        ValueType value = 0.0;

        int pos = ii;  // position in jdsJA, jdsValues

        int ni  = jdsIlg[ii];  // number entries in row

        for ( int jj = 0; jj < ni; ++jj )
        {
            IndexType j = jdsJA[pos];
            value += jdsValues[pos] * fetch_JDSx<ValueType,useTexture>( x_d, j );
            pos += fetch_JDSdlg<useTexture,useSharedMem>( jdsDLG, dlg, jj );
        }

        result_d[i] += alpha * value;
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType>
void CUDAJDSUtils::normalGEMV(
    ValueType result[],
    const ValueType alpha,
    const ValueType x[],
    const ValueType beta,
    const ValueType y[],
    const IndexType numRows,
    const IndexType jdsPerm[],
    const IndexType jdsILG[],
    const IndexType ndlg,
    const IndexType jdsDLG[],
    const IndexType jdsJA[],
    const ValueType jdsValues[],
    SyncToken* syncToken )
{
    if ( ( beta == static_cast<ValueType>( 1 ) ) && ( result == y ) )
    {
        // result = alpha * A * x + beta * y ->  result += alpha * A * x

        sparseGEMV( result, alpha, x, numRows, jdsPerm, jdsILG, ndlg, jdsDLG, jdsJA, jdsValues, syncToken );

        return;
    }

    LAMA_REGION( "CUDA.JDS.normalGEMV" )

    LAMA_LOG_INFO( logger, "normalGEMV<" << typeid( ValueType ).name() << ">" 
                           << ", #rows = " << numRows << ", #diags = " << ndlg )

    LAMA_LOG_INFO(
        logger, "alpha = " << alpha << ", x = " << x << ", beta = " << beta << ", y = " << y << ", result = " << result )

    bool useTexture = CUDASettings::useTexture();
    const bool useSharedMem = CUDASettings::useSharedMem(); // maybe optimize later

    if ( syncToken )
    {
        // Not yet supported: unbind Texture after synchronization 

        useTexture = false;
    }

    LAMA_LOG_DEBUG( logger, "useTexture = " << useTexture << ", useSharedMem = " << useSharedMem )

    const int block_size = ( numRows > 8191 ? 256 : 128 );

    dim3 dimBlock( block_size, 1, 1 );
    dim3 dimGrid = makeGrid( numRows, dimBlock.x );

    LAMA_CHECK_CUDA_ACCESS

    hipStream_t stream = 0;  // default stream if no SyncToken is available

    LAMA_LOG_INFO( logger, "Start jdsgemv_kernel<" << typeid( ValueType ).name() 
                           << ", useTexture = " << useTexture << ", useSharedMem = " << useSharedMem << ">" );

    if ( syncToken )
    {
        CUDAStreamSyncToken* cudaStreamSyncToken = dynamic_cast<CUDAStreamSyncToken*>( syncToken );
        LAMA_ASSERT_DEBUG( cudaStreamSyncToken, "no cuda stream sync token provided" )
        stream = cudaStreamSyncToken->getCUDAStream();
    }

    if ( useTexture )
    {
        jdsBindTexture( x );

        if ( useSharedMem )
        {
            const int sharedMemSize = ndlg * sizeof(int);
            hipFuncSetCacheConfig(reinterpret_cast<const void*>( jdsgemvKernel<ValueType), true, true>, hipFuncCachePreferL1 );
            jdsgemvKernel<ValueType, true, true><<<dimGrid, dimBlock, sharedMemSize, stream>>>
            ( numRows, alpha, jdsValues, jdsDLG, ndlg, jdsILG, jdsJA, jdsPerm, x, beta, y, result);
        }
        else // no sharedMem
        {
            LAMA_CUDA_RT_CALL( hipBindTexture( NULL, texJDSdlgRef, jdsDLG ), "LAMA_STATUS_CUDA_BINDTEX_FAILED" );
            hipFuncSetCacheConfig(reinterpret_cast<const void*>( jdsgemvKernel<ValueType), true, false>, hipFuncCachePreferL1 );
            jdsgemvKernel<ValueType, true, false><<<dimGrid, dimBlock, 0, stream>>>
            ( numRows, alpha, jdsValues, jdsDLG, ndlg, jdsILG, jdsJA, jdsPerm, x, beta, y, result);
        }

        // skip the following in case of asynchronous execution 

        if ( !syncToken )
        {
             LAMA_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "JDS: gemvKernel FAILED" )
        }

        if ( !useSharedMem )
        {
            LAMA_CUDA_RT_CALL( hipUnbindTexture( texJDSdlgRef ), "unbind texture for DLG of JDS" );
        }

        jdsUnbindTexture( x );
    }
 
    else  // no Texture cache

    {
        if ( useSharedMem )
        {
            const int sharedMemSize = ndlg * sizeof( int );
            hipFuncSetCacheConfig(reinterpret_cast<const void*>( jdsgemvKernel<ValueType), false, true>, hipFuncCachePreferL1 );
            jdsgemvKernel<ValueType, false, true><<<dimGrid, dimBlock, sharedMemSize, stream>>>
            ( numRows, alpha, jdsValues, jdsDLG, ndlg, jdsILG, jdsJA, jdsPerm, x, beta, y, result);
        }
        else // no sharedMem
        {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>( jdsgemvKernel<ValueType), false, false>, hipFuncCachePreferL1 );
            jdsgemvKernel<ValueType, false, false><<<dimGrid, dimBlock, 0, stream>>>
            ( numRows, alpha, jdsValues, jdsDLG, ndlg, jdsILG, jdsJA, jdsPerm, x, beta, y, result);
        }

        if ( !syncToken )
        {
            LAMA_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "JDS: gemvKernel FAILED" )
        }
    }
}

/* --------------------------------------------------------------------------- */

template<typename ValueType>
void CUDAJDSUtils::sparseGEMV(
    ValueType result[],
    const ValueType alpha,
    const ValueType x[],
    const IndexType numRows,
    const IndexType jdsPerm[],
    const IndexType jdsILG[],
    const IndexType ndlg,
    const IndexType jdsDLG[],
    const IndexType jdsJA[],
    const ValueType jdsValues[],
    SyncToken* syncToken )
{
    LAMA_REGION( "CUDA.JDS.sparseGEMV" )

    LAMA_LOG_INFO( logger, "sparseGEMV<" << typeid( ValueType ).name() << ">" 
                           << ", #rows = " << numRows << ", #diags = " << ndlg )

    if ( ndlg == 0 )
    {
        return;    // nothing to do 
    }

    IndexType nonEmptyRows = numRows;

    LAMA_CUDA_RT_CALL( hipMemcpy( &nonEmptyRows, &jdsDLG[0], sizeof( IndexType ), hipMemcpyDeviceToHost ),
                       "dlg[0] for number of non-empty rows" )

    bool useTexture = CUDASettings::useTexture();
    const bool useSharedMem = CUDASettings::useSharedMem(); // maybe optimize later

    if ( syncToken )
    {
        // Not yet supported: unbind Texture after synchronization 

        useTexture = false;
    }

    LAMA_LOG_DEBUG( logger, "useTexture = " << useTexture << ", useSharedMem = " << useSharedMem )

    const int block_size = ( nonEmptyRows > 8191 ? 256 : 128 );

    dim3 dimBlock( block_size, 1, 1 );
    dim3 dimGrid = makeGrid( nonEmptyRows, dimBlock.x );

    LAMA_CHECK_CUDA_ACCESS

    hipStream_t stream = 0;  // default stream if no SyncToken is available

    LAMA_LOG_INFO( logger, "Start jdsgemv_kernel<" << typeid( ValueType ).name() 
                           << ", useTexture = " << useTexture << ", useSharedMem = " << useSharedMem << ">" );

    if ( syncToken )
    {
        CUDAStreamSyncToken* cudaStreamSyncToken = dynamic_cast<CUDAStreamSyncToken*>( syncToken );
        LAMA_ASSERT_DEBUG( cudaStreamSyncToken, "no cuda stream sync token provided" )
        stream = cudaStreamSyncToken->getCUDAStream();
    }

    if ( useTexture )
    {
        jdsBindTexture( x );

        if ( useSharedMem )
        {
            const int sharedMemSize = ndlg * sizeof(int);
            hipFuncSetCacheConfig(reinterpret_cast<const void*>( jdsgemvSparseKernel<ValueType), true, true>, hipFuncCachePreferL1 );
            jdsgemvSparseKernel<ValueType, true, true><<<dimGrid, dimBlock, sharedMemSize, stream>>>
            ( numRows, alpha, jdsValues, jdsDLG, ndlg, jdsILG, jdsJA, jdsPerm, x, result);
        }
        else // no sharedMem
        {
            LAMA_CUDA_RT_CALL( hipBindTexture( NULL, texJDSdlgRef, jdsDLG ), "LAMA_STATUS_CUDA_BINDTEX_FAILED" );
            hipFuncSetCacheConfig(reinterpret_cast<const void*>( jdsgemvSparseKernel<ValueType), true, false>, hipFuncCachePreferL1 );
            jdsgemvSparseKernel<ValueType, true, false><<<dimGrid, dimBlock, 0, stream>>>
            ( numRows, alpha, jdsValues, jdsDLG, ndlg, jdsILG, jdsJA, jdsPerm, x, result);
        }

        // skip the following in case of asynchronous execution 

        if ( !syncToken )
        {
             LAMA_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "JDS: gemvSparseKernel FAILED" )
        }

        if ( !useSharedMem )
        {
            LAMA_CUDA_RT_CALL( hipUnbindTexture( texJDSdlgRef ), "unbind texture for DLG of JDS" );
        }

        jdsUnbindTexture( x );
    }
 
    else  // no Texture cache

    {
        if ( useSharedMem )
        {
            const int sharedMemSize = ndlg * sizeof( int );
            hipFuncSetCacheConfig(reinterpret_cast<const void*>( jdsgemvSparseKernel<ValueType), false, true>, hipFuncCachePreferL1 );
            jdsgemvSparseKernel<ValueType, false, true><<<dimGrid, dimBlock, sharedMemSize, stream>>>
            ( numRows, alpha, jdsValues, jdsDLG, ndlg, jdsILG, jdsJA, jdsPerm, x, result);
        }
        else // no sharedMem
        {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>( jdsgemvSparseKernel<ValueType), false, false>, hipFuncCachePreferL1 );
            jdsgemvSparseKernel<ValueType, false, false><<<dimGrid, dimBlock, 0, stream>>>
            ( numRows, alpha, jdsValues, jdsDLG, ndlg, jdsILG, jdsJA, jdsPerm, x, result);
        }

        if ( !syncToken )
        {
            LAMA_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "JDS: gemvSparseKernel FAILED" )
        }
    }
}

/* --------------------------------------------------------------------------- */

void CUDAJDSUtils::setInterface( JDSUtilsInterface& JDSUtils )
{
    LAMA_LOG_INFO( logger, "set JDS routines for CUDA in Interface" )

    LAMA_INTERFACE_REGISTER( JDSUtils, sortRows )
    LAMA_INTERFACE_REGISTER( JDSUtils, checkDiagonalProperty )
    LAMA_INTERFACE_REGISTER( JDSUtils, ilg2dlg )
    LAMA_INTERFACE_REGISTER( JDSUtils, setInversePerm )

    LAMA_INTERFACE_REGISTER_TT( JDSUtils, scaleValue, float, float )
    LAMA_INTERFACE_REGISTER_TT( JDSUtils, scaleValue, float, double )
    LAMA_INTERFACE_REGISTER_TT( JDSUtils, scaleValue, double, float )
    LAMA_INTERFACE_REGISTER_TT( JDSUtils, scaleValue, double, double )

    LAMA_INTERFACE_REGISTER_TT( JDSUtils, getRow, float, float )
    LAMA_INTERFACE_REGISTER_TT( JDSUtils, getRow, float, double )
    LAMA_INTERFACE_REGISTER_TT( JDSUtils, getRow, double, float )
    LAMA_INTERFACE_REGISTER_TT( JDSUtils, getRow, double, double )

    LAMA_INTERFACE_REGISTER_TT( JDSUtils, getValue, float, float )
    LAMA_INTERFACE_REGISTER_TT( JDSUtils, getValue, float, double )
    LAMA_INTERFACE_REGISTER_TT( JDSUtils, getValue, double, float )
    LAMA_INTERFACE_REGISTER_TT( JDSUtils, getValue, double, double )

    LAMA_INTERFACE_REGISTER_TT( JDSUtils, getCSRValues, float, float )
    LAMA_INTERFACE_REGISTER_TT( JDSUtils, getCSRValues, float, double )
    LAMA_INTERFACE_REGISTER_TT( JDSUtils, getCSRValues, double, float )
    LAMA_INTERFACE_REGISTER_TT( JDSUtils, getCSRValues, double, double )

    LAMA_INTERFACE_REGISTER_TT( JDSUtils, setCSRValues, float, float )
    LAMA_INTERFACE_REGISTER_TT( JDSUtils, setCSRValues, float, double )
    LAMA_INTERFACE_REGISTER_TT( JDSUtils, setCSRValues, double, float )
    LAMA_INTERFACE_REGISTER_TT( JDSUtils, setCSRValues, double, double )

    LAMA_INTERFACE_REGISTER_T( JDSUtils, jacobi, float )
    LAMA_INTERFACE_REGISTER_T( JDSUtils, jacobi, double )

    LAMA_INTERFACE_REGISTER_T( JDSUtils, normalGEMV, float )
    LAMA_INTERFACE_REGISTER_T( JDSUtils, normalGEMV, double )

    LAMA_INTERFACE_REGISTER_T( JDSUtils, jacobiHalo, float )
    LAMA_INTERFACE_REGISTER_T( JDSUtils, jacobiHalo, double )
}

/* --------------------------------------------------------------------------- */
/*    Static registration of the Utils routines                                */
/* --------------------------------------------------------------------------- */

bool CUDAJDSUtils::registerInterface()
{
    LAMAInterface& interface = LAMAInterfaceRegistry::getRegistry().modifyInterface( Context::CUDA );
    setInterface( interface.JDSUtils );
    return true;
}

/* --------------------------------------------------------------------------- */
/*    Static initialiazion at program start                                    */
/* --------------------------------------------------------------------------- */

bool CUDAJDSUtils::initialized = registerInterface();


} // namespace lama
