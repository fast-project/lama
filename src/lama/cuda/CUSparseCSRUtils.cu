/**
 * @file CUSparseCSRUtils.cu
 *
 * @license
 * Copyright (c) 2009-2013
 * Fraunhofer Institute for Algorithms and Scientific Computing SCAI
 * for Fraunhofer-Gesellschaft
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 * @endlicense
 *
 * @brief Implementation of some CSR routines with CUSparse library 5.0
 * @author Bea Hornef, Thomas Brandes, Jiri Kraus
 * @date 04.07.2012
 * @since 1.0.0
 */

#include <lama/LAMAInterface.hpp>
#include <lama/LAMAInterfaceRegistry.hpp>

#include <lama/cuda/utils.cu.h>
#include <lama/cuda/CUDAError.hpp>
#include <lama/cuda/CUSparseCSRUtils.hpp>

#include <hip/hip_runtime.h>
#include <hipsparse.h>

#include <lama/tracing.hpp>

#include <lama/ContextFactory.hpp>

namespace lama
{

LAMA_LOG_DEF_LOGGER( CUSparseCSRUtils::logger, "CUDA.CSRUtilsSparse" )

/* --------------------------------------------------------------------------- */
/*     cusparse handle is needed, set by HIPContext                           */
/* --------------------------------------------------------------------------- */

extern hipsparseHandle_t CUDAContext_cusparseHandle;

/* --------------------------------------------------------------------------- */
/*     Template specialization convertCSR2CSC<float>                           */
/* --------------------------------------------------------------------------- */

template<>
void CUSparseCSRUtils::convertCSR2CSC(
    IndexType cscIA[],
    IndexType cscJA[],
    float cscValues[],
    const IndexType csrIA[],
    const IndexType csrJA[],
    const float csrValues[],
    int numRows,
    int numColumns,
    int /* numValues */)
{
    LAMA_LOG_INFO( logger,
                   "convertCSR2CSC<float> -> hipsparseScsr2csc" << ", matrix size = " << numRows << " x " << numColumns )

    int numValues = 0;

    LAMA_CUSPARSE_CALL(
        hipsparseScsr2csc( CUDAContext_cusparseHandle, 
                          numRows, numColumns, numValues,
                          csrValues, csrIA, csrJA, 
                          cscValues, cscJA, cscIA, 
                          HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO ),
        "convertCSR2SCC<float>" )

    LAMA_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "convertCSR2CSC" )
}

/* --------------------------------------------------------------------------- */
/*     Template specialization convertCSR2CSC<double>                          */
/* --------------------------------------------------------------------------- */

template<>
void CUSparseCSRUtils::convertCSR2CSC(
    IndexType cscIA[],
    IndexType cscJA[],
    double cscValues[],
    const IndexType csrIA[],
    const IndexType csrJA[],
    const double csrValues[],
    int numRows,
    int numColumns,
    int /* numValues */)
{
    LAMA_LOG_INFO( logger,
                   "convertCSR2CSC<double> -> hipsparseDcsr2csc" << ", matrix size = " << numRows << " x " << numColumns )

    int numValues = 0;

    LAMA_CUSPARSE_CALL(
        hipsparseDcsr2csc( CUDAContext_cusparseHandle, 
                          numRows, numColumns, numValues,
                          csrValues, csrIA, csrJA, 
                          cscValues, cscJA, cscIA, 
                          HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO ),
        "convertCSR2SCC<double>" )

    LAMA_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "convertCSR2CSC" )
}

/* ------------------------------------------------------------------------------------------------------------------ */
/*                                             matrixAddSizes                                                         */
/* ------------------------------------------------------------------------------------------------------------------ */

IndexType CUSparseCSRUtils::matrixAddSizes(
    IndexType cIA[],
    const IndexType numRows,
    const IndexType numColumns,
    bool diagonalProperty,
    const IndexType aIA[],
    const IndexType aJA[],
    const IndexType bIA[],
    const IndexType bJA[] )
{
    LAMA_REGION( "CUDA.CSR.matrixAddSizes" )

    LAMA_LOG_INFO(
        logger,
        "matrixAddSizes for " << numRows << " x " << numColumns << " matrix" << ", diagonalProperty = " << diagonalProperty )

    LAMA_CHECK_CUDA_ACCESS

    hipsparseMatDescr_t descrCSR;

    LAMA_CUSPARSE_CALL( hipsparseCreateMatDescr( &descrCSR ), "hipsparseCreateMatDescr" )

    hipsparseSetMatType( descrCSR, HIPSPARSE_MATRIX_TYPE_GENERAL );
    hipsparseSetMatIndexBase( descrCSR, HIPSPARSE_INDEX_BASE_ZERO );  

    int nnzA = 0;  // aIA[ m ]
    int nnzB = 0;  // bIA[ numColumns ]

    // we have not passed the values, so copy it from device to host

    hipMemcpy( &nnzA, &aIA[numRows], 1, hipMemcpyDeviceToHost );
    hipMemcpy( &nnzB, &bIA[numRows], 1, hipMemcpyDeviceToHost );

    int nnzC;

    LAMA_CUSPARSE_CALL(
        hipsparseXcsrgeamNnz( CUDAContext_cusparseHandle, 
                             numRows, numColumns,
                             descrCSR, nnzA, aIA, aJA,
                             descrCSR, nnzB, bIA, bJA,
                             descrCSR, cIA, &nnzC ), 
        "hipsparseXcsrgeamNnz" )

    // synchronization might be redundant due to the return value

    LAMA_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "hipsparseXcsrgeamNnz" )

    return nnzC;
}

/* ------------------------------------------------------------------------------------------------------------------ */
/*                                             matrixMultiplySizes                                                    */
/* ------------------------------------------------------------------------------------------------------------------ */

IndexType CUSparseCSRUtils::matrixMultiplySizes(
    IndexType cIA[],
    const IndexType m,
    const IndexType n,
    const IndexType k,
    bool diagonalProperty,
    const IndexType aIA[],
    const IndexType aJA[],
    const IndexType bIA[],
    const IndexType bJA[] )
{
    LAMA_REGION( "CUDA.CSR.matrixMultiplySizes" )

    LAMA_LOG_ERROR(
        logger,
        "matrixMutliplySizes for " << m << " x " << n << " matrix" << ", diagonalProperty = " << diagonalProperty )

    LAMA_CHECK_CUDA_ACCESS

    hipsparseMatDescr_t descrCSR;

    LAMA_CUSPARSE_CALL( hipsparseCreateMatDescr( &descrCSR ), "hipsparseCreateMatDescr" )

    hipsparseSetMatType( descrCSR, HIPSPARSE_MATRIX_TYPE_GENERAL );
    hipsparseSetMatIndexBase( descrCSR, HIPSPARSE_INDEX_BASE_ZERO );  

    int nnzA = 0;  // aIA[ m ]
    int nnzB = 0;  // bIA[ numColumns ]

    // we have not passed the values, so copy it

    hipMemcpy( &nnzA, &aIA[m], 1, hipMemcpyDeviceToHost );
    hipMemcpy( &nnzB, &bIA[k], 1, hipMemcpyDeviceToHost );

    int nnzC;

    LAMA_CUSPARSE_CALL(
        hipsparseXcsrgemmNnz( CUDAContext_cusparseHandle, 
                             HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
                             m, n, k, 
                             descrCSR, nnzA, aIA, aJA,
                             descrCSR, nnzB, bIA, bJA,
                             descrCSR, cIA, &nnzC ), 
        "hipsparseXcsrgemmNnz" )

    // synchronization might be redundant due to the return value

    LAMA_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "convertCSR2CSC" )

    return nnzC;
}

/* ------------------------------------------------------------------------------------------------------------------ */
/*                                             matrixAdd                                                              */
/* ------------------------------------------------------------------------------------------------------------------ */

template<>
void CUSparseCSRUtils::matrixAdd(
    IndexType cJA[],
    float cValues[],
    const IndexType cIA[],
    const IndexType numRows,
    const IndexType numColumns,
    bool diagonalProperty,
    const float alpha,
    const IndexType aIA[],
    const IndexType aJA[],
    const float aValues[],
    const float beta,
    const IndexType bIA[],
    const IndexType bJA[],
    const float bValues[] )
{
    LAMA_REGION( "CUDA.CSR.matrixAdd" )

    LAMA_LOG_INFO( logger, "matrixAdd for " << numRows << "x" << numColumns << " matrix" )

    LAMA_CHECK_CUDA_ACCESS

    hipsparseMatDescr_t descrCSR;

    LAMA_CUSPARSE_CALL( hipsparseCreateMatDescr( &descrCSR ), "hipsparseCreateMatDescr" )

    hipsparseSetMatType( descrCSR, HIPSPARSE_MATRIX_TYPE_GENERAL );
    hipsparseSetMatIndexBase( descrCSR, HIPSPARSE_INDEX_BASE_ZERO );  

    int nnzA = 0;  // aIA[ m ]
    int nnzB = 0;  // bIA[ numColumns ]

    // we have not passed the values, so copy it

    hipMemcpy( &nnzA, &aIA[numRows], 1, hipMemcpyDeviceToHost );
    hipMemcpy( &nnzB, &bIA[numRows], 1, hipMemcpyDeviceToHost );

    // cIA requires const_cast, but will not be modified

    LAMA_CUSPARSE_CALL(
        hipsparseScsrgeam( CUDAContext_cusparseHandle, 
                          numRows, numColumns, 
                          &alpha, descrCSR, nnzA, aValues, aIA, aJA,
                          &beta, descrCSR, nnzB, bValues, bIA, bJA,
                          descrCSR, cValues, const_cast<IndexType*>( cIA ), cJA ), 
        "hipsparseScsrgeam" )

    // synchronization might be redundant

    LAMA_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "hipsparseScsrgeam" )
}

template<>
void CUSparseCSRUtils::matrixAdd(
    IndexType cJA[],
    double cValues[],
    const IndexType cIA[],
    const IndexType numRows,
    const IndexType numColumns,
    bool diagonalProperty,
    const double alpha,
    const IndexType aIA[],
    const IndexType aJA[],
    const double aValues[],
    const double beta,
    const IndexType bIA[],
    const IndexType bJA[],
    const double bValues[] )
{
    LAMA_REGION( "CUDA.CSR.matrixAdd" )

    LAMA_LOG_INFO( logger, "matrixAdd for " << numRows << "x" << numColumns << " matrix" )

    LAMA_CHECK_CUDA_ACCESS

    hipsparseMatDescr_t descrCSR;

    LAMA_CUSPARSE_CALL( hipsparseCreateMatDescr( &descrCSR ), "hipsparseCreateMatDescr" )

    hipsparseSetMatType( descrCSR, HIPSPARSE_MATRIX_TYPE_GENERAL );
    hipsparseSetMatIndexBase( descrCSR, HIPSPARSE_INDEX_BASE_ZERO );  

    int nnzA = 0;  // aIA[ numRows ]
    int nnzB = 0;  // bIA[ numColumns ]

    // we have not passed the number of non-zero values for A, B, so copy it

    hipMemcpy( &nnzA, &aIA[numRows], 1, hipMemcpyDeviceToHost );
    hipMemcpy( &nnzB, &bIA[numRows], 1, hipMemcpyDeviceToHost );

    // cIA requires const_cast, but will not be modified

    LAMA_CUSPARSE_CALL(
        hipsparseDcsrgeam( CUDAContext_cusparseHandle, 
                          numRows, numColumns,
                          &alpha, descrCSR, nnzA, aValues, aIA, aJA,
                          &beta, descrCSR, nnzB, bValues, bIA, bJA,
                          descrCSR, cValues, const_cast<IndexType*>( cIA ), cJA ), 
        "hipsparseDcsrgeam" )

    // synchronization might be redundant

    LAMA_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "hipsparseDcsrgeam" )
}

/* ------------------------------------------------------------------------------------------------------------------ */
/*                                             matrixMultiply                                                         */
/* ------------------------------------------------------------------------------------------------------------------ */

template<>
void CUSparseCSRUtils::matrixMultiply(
    const IndexType cIA[],
    IndexType cJA[],
    float cValues[],
    const IndexType m,
    const IndexType n,
    const IndexType k,
    const float alpha,
    bool diagonalProperty,
    const IndexType aIA[],
    const IndexType aJA[],
    const float aValues[],
    const IndexType bIA[],
    const IndexType bJA[],
    const float bValues[] )
{
    LAMA_REGION( "CUDA.CSR.matrixMultiply" )

    LAMA_LOG_INFO( logger, "matrixMultiply, result is " << m << "x" << n << " CSR storage" )

    LAMA_CHECK_CUDA_ACCESS

    hipsparseMatDescr_t descrCSR;

    LAMA_CUSPARSE_CALL( hipsparseCreateMatDescr( &descrCSR ), "hipsparseCreateMatDescr" )

    hipsparseSetMatType( descrCSR, HIPSPARSE_MATRIX_TYPE_GENERAL );
    hipsparseSetMatIndexBase( descrCSR, HIPSPARSE_INDEX_BASE_ZERO );  

    int nnzA = 0;  // aIA[ m ]
    int nnzB = 0;  // bIA[ numColumns ]

    // we have not passed the number of non-zero values for A, B, so copy it

    hipMemcpy( &nnzA, &aIA[m], 1, hipMemcpyDeviceToHost );
    hipMemcpy( &nnzB, &bIA[k], 1, hipMemcpyDeviceToHost );

    LAMA_ASSERT_EQUAL_ERROR( static_cast<float>( 1 ), alpha );

    LAMA_CUSPARSE_CALL(
        hipsparseScsrgemm( CUDAContext_cusparseHandle, 
                          HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
                          m, n, k, 
                          descrCSR, nnzA, aValues, aIA, aJA,
                          descrCSR, nnzB, bValues, bIA, bJA,
                          descrCSR, cValues, cIA, cJA ), 
        "hipsparseScsrgemm" )

    // synchronization might be redundant d

    LAMA_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "csrSparseMatmulS" )
}

template<>
void CUSparseCSRUtils::matrixMultiply(
    const IndexType cIA[],
    IndexType cJA[],
    double cValues[],
    const IndexType m,
    const IndexType n,
    const IndexType k,
    const double alpha,
    bool diagonalProperty,
    const IndexType aIA[],
    const IndexType aJA[],
    const double aValues[],
    const IndexType bIA[],
    const IndexType bJA[],
    const double bValues[] )
{
    LAMA_REGION( "CUDA.CSR.matrixMultiply" )

    LAMA_LOG_INFO( logger, "matrixMultiply, result is " << m << "x" << n << " CSR storage" )

    LAMA_CHECK_CUDA_ACCESS

    hipsparseMatDescr_t descrCSR;

    LAMA_CUSPARSE_CALL( hipsparseCreateMatDescr( &descrCSR ), "hipsparseCreateMatDescr" )

    hipsparseSetMatType( descrCSR, HIPSPARSE_MATRIX_TYPE_GENERAL );
    hipsparseSetMatIndexBase( descrCSR, HIPSPARSE_INDEX_BASE_ZERO );  

    int nnzA = 0;  // aIA[ m ]
    int nnzB = 0;  // bIA[ n ]

    // we have not passed the number of non-zero values for A, B, so copy it

    hipMemcpy( &nnzA, &aIA[m], 1, hipMemcpyDeviceToHost );
    hipMemcpy( &nnzB, &bIA[k], 1, hipMemcpyDeviceToHost );

    LAMA_ASSERT_EQUAL_ERROR( static_cast<double>( 1 ), alpha );

    LAMA_CUSPARSE_CALL(
        hipsparseDcsrgemm( CUDAContext_cusparseHandle, 
                          HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
                          m, n, k, 
                          descrCSR, nnzA, aValues, aIA, aJA,
                          descrCSR, nnzB, bValues, bIA, bJA,
                          descrCSR, cValues, cIA, cJA ), 
        "hipsparseDcsrgemm" )

    // synchronization might be redundant d

    LAMA_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "csrSparseMatmulD" )
}

/* ------------------------------------------------------------------------------------------------------------------ */

/* --------------------------------------------------------------------------- */
/*     Template instantiations via registration routine                        */
/* --------------------------------------------------------------------------- */

void CUSparseCSRUtils::setInterface( CSRUtilsInterface& CSRUtils )
{
    LAMA_LOG_INFO( logger, "set CSR routines for CUSparse in Interface" )

    LAMA_INTERFACE_REGISTER_T( CSRUtils, convertCSR2CSC, float )
    LAMA_INTERFACE_REGISTER_T( CSRUtils, convertCSR2CSC, double )

    LAMA_INTERFACE_REGISTER( CSRUtils, matrixAddSizes )
    LAMA_INTERFACE_REGISTER( CSRUtils, matrixMultiplySizes )

    LAMA_INTERFACE_REGISTER_T( CSRUtils, matrixAdd, float )
    LAMA_INTERFACE_REGISTER_T( CSRUtils, matrixAdd, double )

    LAMA_INTERFACE_REGISTER_T( CSRUtils, matrixMultiply, float )
    LAMA_INTERFACE_REGISTER_T( CSRUtils, matrixMultiply, double )
}

/* --------------------------------------------------------------------------- */
/*    Static registration of the Utils routines                                */
/* --------------------------------------------------------------------------- */

bool CUSparseCSRUtils::registerInterface()
{
    LAMAInterface& interface = LAMAInterfaceRegistry::getRegistry().modifyInterface( Context::CUDA );
    setInterface( interface.CSRUtils );
    return true;
}

/* --------------------------------------------------------------------------- */
/*    Static initialiazion at program start                                    */
/* --------------------------------------------------------------------------- */

bool CUSparseCSRUtils::initialized = registerInterface();

} // namespace lama
