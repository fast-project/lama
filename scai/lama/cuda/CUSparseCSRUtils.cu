/**
 * @file CUSparseCSRUtils.cu
 *
 * @license
 * Copyright (c) 2009-2013
 * Fraunhofer Institute for Algorithms and Scientific Computing SCAI
 * for Fraunhofer-Gesellschaft
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 * @endlicense
 *
 * @brief Implementation of some CSR routines with CUSparse library 5.0
 * @author Thomas Brandes
 * @date 11.06.2013
 * @since 1.0.1
 */

#include <scai/lama/LAMAInterface.hpp>
#include <scai/lama/LAMAInterfaceRegistry.hpp>

#include <scai/lama/cuda/utils.cu.h>
#include <scai/lama/cuda/CUDAError.hpp>
#include <scai/lama/cuda/CUSparseCSRUtils.hpp>
#include <scai/memory/cuda/CUDAStreamSyncToken.hpp>
#include <scai/lama/Settings.hpp>

#include <hip/hip_runtime.h>
#include <hipsparse.h>

#include <scai/tracing.hpp>

#include <scai/lama/ContextFactory.hpp>

namespace lama
{

    SCAI_LOG_DEF_LOGGER( CUSparseCSRUtils::logger, "CUDA.CSRUtilsSparse" )

    /* --------------------------------------------------------------------------- */
    /*     cusparse handle is needed, set by HIPContext                           */
    /* --------------------------------------------------------------------------- */

    extern hipsparseHandle_t CUDAContext_cusparseHandle;

    /* --------------------------------------------------------------------------- */
    /*     Template specialization convertCSR2CSC<float>                           */
    /* --------------------------------------------------------------------------- */

    template<>
    void CUSparseCSRUtils::convertCSR2CSC(
                    IndexType cscIA[],
                    IndexType cscJA[],
                    float cscValues[],
                    const IndexType csrIA[],
                    const IndexType csrJA[],
                    const float csrValues[],
                    int numRows,
                    int numColumns,
                    int numValues )
    {
        SCAI_LOG_INFO( logger,
                        "convertCSR2CSC<float> -> hipsparseScsr2csc" << ", matrix size = "
                        << numRows << " x " << numColumns << ", nnz = " << numValues )

        LAMA_CUSPARSE_CALL(
                        hipsparseScsr2csc( CUDAContext_cusparseHandle,
                                        numRows, numColumns, numValues,
                                        csrValues, csrIA, csrJA,
                                        cscValues, cscJA, cscIA,
                                        HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO ),
                        "convertCSR2SCC<float>" )

        LAMA_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "convertCSR2CSC" )
    }

    /* --------------------------------------------------------------------------- */
    /*     Template specialization convertCSR2CSC<double>                          */
    /* --------------------------------------------------------------------------- */

    template<>
    void CUSparseCSRUtils::convertCSR2CSC(
                    IndexType cscIA[],
                    IndexType cscJA[],
                    double cscValues[],
                    const IndexType csrIA[],
                    const IndexType csrJA[],
                    const double csrValues[],
                    int numRows,
                    int numColumns,
                    int numValues )
    {
        SCAI_LOG_INFO( logger,
                        "convertCSR2CSC<double> -> hipsparseDcsr2csc" << ", matrix size = "
                        << numRows << " x " << numColumns << ", nnz = " << numValues )

        LAMA_CUSPARSE_CALL(
                        hipsparseDcsr2csc( CUDAContext_cusparseHandle,
                                        numRows, numColumns, numValues,
                                        csrValues, csrIA, csrJA,
                                        cscValues, cscJA, cscIA,
                                        HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO ),
                        "convertCSR2SCC<double>" )

        LAMA_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "convertCSR2CSC" )
    }

    /* ------------------------------------------------------------------------------------------------------------------ */
    /*                                             normalGEMV                                                             */
    /* ------------------------------------------------------------------------------------------------------------------ */

    template<>
    void CUSparseCSRUtils::normalGEMV(
                    float result[],
                    const float alpha,
                    const float x[],
                    const float beta,
                    const float y[],
                    const IndexType numRows,
                    const IndexType numColumns,
                    const IndexType nnz,
                    const IndexType csrIA[],
                    const IndexType csrJA[],
                    const float csrValues[],
                    SyncToken* syncToken )
    {
        SCAI_LOG_INFO( logger, "normalGEMV<float>" <<
                        " result[ " << numRows << "] = " << alpha << " * A(csr) * x + " << beta << " * y " )

        SCAI_LOG_DEBUG( logger, "x = " << x << ", y = " << y << ", result = " << result )

        LAMA_CHECK_CUDA_ACCESS

        hipStream_t stream = 0; // default stream if no syncToken is given

        hipsparseMatDescr_t descrCSR;

        LAMA_CUSPARSE_CALL( hipsparseCreateMatDescr( &descrCSR ), "hipsparseCreateMatDescr" )

        hipsparseSetMatType( descrCSR, HIPSPARSE_MATRIX_TYPE_GENERAL );
        hipsparseSetMatIndexBase( descrCSR, HIPSPARSE_INDEX_BASE_ZERO );

        if ( syncToken )
        {
            CUDAStreamSyncToken* cudaStreamSyncToken = dynamic_cast<CUDAStreamSyncToken*>( syncToken );
            LAMA_ASSERT_DEBUG( cudaStreamSyncToken, "no cuda stream sync token provided" )
            stream = cudaStreamSyncToken->getCUDAStream();
            LAMA_CUSPARSE_CALL( hipsparseSetStream( CUDAContext_cusparseHandle, stream ),
                            "hipsparseSetStream" )
        }

        if ( y != result && beta != 0 )
        {
            LAMA_CUDA_RT_CALL( hipMemcpy( result, y, numRows * sizeof( float ), hipMemcpyDeviceToDevice ),
                            "hipMemcpy for result = y" )
        }

        // call result = alpha * op(A) * x + beta * result of cusparse
        // Note: alpha, beta are passed as pointers

        SCAI_LOG_INFO( logger, "Start hipsparseScsrmv, stream = " << stream )

        LAMA_CUSPARSE_CALL( hipsparseScsrmv( CUDAContext_cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                        numRows, numColumns, nnz, &alpha, descrCSR,
                                        csrValues, csrIA, csrJA, x, &beta, result ),
                        "hipsparseScsrmv" )

        if ( syncToken )
        {
            // set back stream for cusparse

            LAMA_CUSPARSE_CALL( hipsparseSetStream( CUDAContext_cusparseHandle, 0 ),
                            "hipsparseSetStream" )
        }
        else
        {
            LAMA_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "hipsparseXcsrgeamNnz" )
        }
    }

    template<>
    void CUSparseCSRUtils::normalGEMV(
                    double result[],
                    const double alpha,
                    const double x[],
                    const double beta,
                    const double y[],
                    const IndexType numRows,
                    const IndexType numColumns,
                    const IndexType nnz,
                    const IndexType csrIA[],
                    const IndexType csrJA[],
                    const double csrValues[],
                    SyncToken* syncToken )
    {
        SCAI_LOG_INFO( logger, "normalGEMV<double>" <<
                        " result[ " << numRows << "] = " << alpha << " * A(csr) * x + " << beta << " * y " )

        SCAI_LOG_DEBUG( logger, "x = " << x << ", y = " << y << ", result = " << result )

        LAMA_CHECK_CUDA_ACCESS

        hipStream_t stream = 0; // default stream if no syncToken is given

        hipsparseMatDescr_t descrCSR;

        LAMA_CUSPARSE_CALL( hipsparseCreateMatDescr( &descrCSR ), "hipsparseCreateMatDescr" )

        hipsparseSetMatType( descrCSR, HIPSPARSE_MATRIX_TYPE_GENERAL );
        hipsparseSetMatIndexBase( descrCSR, HIPSPARSE_INDEX_BASE_ZERO );

        if ( syncToken )
        {
            CUDAStreamSyncToken* cudaStreamSyncToken = dynamic_cast<CUDAStreamSyncToken*>( syncToken );
            LAMA_ASSERT_DEBUG( cudaStreamSyncToken, "no cuda stream sync token provided" )
            stream = cudaStreamSyncToken->getCUDAStream();
            LAMA_CUSPARSE_CALL( hipsparseSetStream( CUDAContext_cusparseHandle, stream ),
                            "hipsparseSetStream" )
        }

        if ( y != result && beta != 0 )
        {
            LAMA_CUDA_RT_CALL( hipMemcpy( result, y, numRows * sizeof( double ), hipMemcpyDeviceToDevice ),
                            "hipMemcpy for result = y" )
        }

        // call result = alpha * op(A) * x + beta * result of cusparse
        // Note: alpha, beta are passed as pointers

        SCAI_LOG_INFO( logger, "Start hipsparseDcsrmv, stream = " << stream )

        LAMA_CUSPARSE_CALL( hipsparseDcsrmv( CUDAContext_cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                        numRows, numColumns, nnz, &alpha, descrCSR,
                                        csrValues, csrIA, csrJA, x, &beta, result ),
                        "hipsparseScsrmv" )

        if ( syncToken )
        {
            // set back stream for cusparse

            LAMA_CUSPARSE_CALL( hipsparseSetStream( CUDAContext_cusparseHandle, 0 ),
                            "hipsparseSetStream" )
        }
        else
        {
            LAMA_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "hipsparseXcsrgeamNnz" )
        }
    }

    /* ------------------------------------------------------------------------------------------------------------------ */
    /*                                             matrixAddSizes                                                         */
    /* ------------------------------------------------------------------------------------------------------------------ */

    IndexType CUSparseCSRUtils::matrixAddSizes(
                    IndexType cIA[],
                    const IndexType numRows,
                    const IndexType numColumns,
                    bool diagonalProperty,
                    const IndexType aIA[],
                    const IndexType aJA[],
                    const IndexType bIA[],
                    const IndexType bJA[] )
    {
        LAMA_REGION( "CUDA.CSR.matrixAddSizes" )

        SCAI_LOG_INFO(
                        logger,
                        "matrixAddSizes for " << numRows << " x " << numColumns << " matrix" << ", diagonalProperty = " << diagonalProperty )

        LAMA_CHECK_CUDA_ACCESS

        hipsparseMatDescr_t descrCSR;

        LAMA_CUSPARSE_CALL( hipsparseCreateMatDescr( &descrCSR ), "hipsparseCreateMatDescr" )

        hipsparseSetMatType( descrCSR, HIPSPARSE_MATRIX_TYPE_GENERAL );
        hipsparseSetMatIndexBase( descrCSR, HIPSPARSE_INDEX_BASE_ZERO );

        int nnzA = 0; // aIA[ m ]
        int nnzB = 0;// bIA[ numColumns ]

        // we have not passed the values, so copy it from device to host

        hipMemcpy( &nnzA, &aIA[numRows], sizeof( IndexType ), hipMemcpyDeviceToHost );
        hipMemcpy( &nnzB, &bIA[numRows], sizeof( IndexType ), hipMemcpyDeviceToHost );

        int nnzC;

        LAMA_CUSPARSE_CALL(
                        hipsparseXcsrgeamNnz( CUDAContext_cusparseHandle,
                                        numRows, numColumns,
                                        descrCSR, nnzA, aIA, aJA,
                                        descrCSR, nnzB, bIA, bJA,
                                        descrCSR, cIA, &nnzC ),
                        "hipsparseXcsrgeamNnz" )

        // synchronization might be redundant due to the return value

        LAMA_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "hipsparseXcsrgeamNnz" )

        return nnzC;
    }

    /* ------------------------------------------------------------------------------------------------------------------ */
    /*                                             matrixMultiplySizes                                                    */
    /* ------------------------------------------------------------------------------------------------------------------ */

    IndexType CUSparseCSRUtils::matrixMultiplySizes(
                    IndexType cIA[],
                    const IndexType m,
                    const IndexType n,
                    const IndexType k,
                    bool diagonalProperty,
                    const IndexType aIA[],
                    const IndexType aJA[],
                    const IndexType bIA[],
                    const IndexType bJA[] )
    {
        LAMA_REGION( "CUDA.CSR.matrixMultiplySizes" )

        SCAI_LOG_INFO(
                        logger,
                        "matrixMutliplySizes for " << m << " x " << n << " matrix" << ", diagonalProperty = " << diagonalProperty )

        LAMA_CHECK_CUDA_ACCESS

        hipsparseMatDescr_t descrCSR;

        LAMA_CUSPARSE_CALL( hipsparseCreateMatDescr( &descrCSR ), "hipsparseCreateMatDescr" )

        hipsparseSetMatType( descrCSR, HIPSPARSE_MATRIX_TYPE_GENERAL );
        hipsparseSetMatIndexBase( descrCSR, HIPSPARSE_INDEX_BASE_ZERO );

        int nnzA = 0; // aIA[ m ]
        int nnzB = 0;// bIA[ numColumns ]

        // we have not passed the values, so copy it

        hipMemcpy( &nnzA, &aIA[m], sizeof( IndexType ), hipMemcpyDeviceToHost );
        hipMemcpy( &nnzB, &bIA[k], sizeof( IndexType ), hipMemcpyDeviceToHost );

        int nnzC;

        SCAI_LOG_DEBUG( logger, "multSizes, A is " << m << " x " << k << ", nnz = " << nnzA
                        << ", B is " << k << " x " << n << ", nnz = " << nnzB
                        << ", C = " << m << " x " << n )

        LAMA_CUSPARSE_CALL(
                        hipsparseXcsrgemmNnz( CUDAContext_cusparseHandle,
                                        HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                        m, n, k,
                                        descrCSR, nnzA, aIA, aJA,
                                        descrCSR, nnzB, bIA, bJA,
                                        descrCSR, cIA, &nnzC ),
                        "hipsparseXcsrgemmNnz" )

        // synchronization might be redundant due to the return value

        LAMA_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "convertCSR2CSC" )

        SCAI_LOG_DEBUG( logger, "matrixMultiplySizes, nnzC = " << nnzC )

        return nnzC;
    }

    /* ------------------------------------------------------------------------------------------------------------------ */
    /*                                             matrixAdd                                                              */
    /* ------------------------------------------------------------------------------------------------------------------ */

    template<>
    void CUSparseCSRUtils::matrixAdd(
                    IndexType cJA[],
                    float cValues[],
                    const IndexType cIA[],
                    const IndexType numRows,
                    const IndexType numColumns,
                    bool diagonalProperty,
                    const float alpha,
                    const IndexType aIA[],
                    const IndexType aJA[],
                    const float aValues[],
                    const float beta,
                    const IndexType bIA[],
                    const IndexType bJA[],
                    const float bValues[] )
    {
        LAMA_REGION( "CUDA.CSR.matrixAdd" )

        SCAI_LOG_INFO( logger, "matrixAdd for " << numRows << "x" << numColumns << " matrix" )

        LAMA_CHECK_CUDA_ACCESS

        hipsparseMatDescr_t descrCSR;

        LAMA_CUSPARSE_CALL( hipsparseCreateMatDescr( &descrCSR ), "hipsparseCreateMatDescr" )

        hipsparseSetMatType( descrCSR, HIPSPARSE_MATRIX_TYPE_GENERAL );
        hipsparseSetMatIndexBase( descrCSR, HIPSPARSE_INDEX_BASE_ZERO );

        int nnzA = 0; // aIA[ m ]
        int nnzB = 0;// bIA[ numColumns ]

        // we have not passed the values, so copy it

        hipMemcpy( &nnzA, &aIA[numRows], sizeof( IndexType ), hipMemcpyDeviceToHost );
        hipMemcpy( &nnzB, &bIA[numRows], sizeof( IndexType ), hipMemcpyDeviceToHost );

        // cIA requires const_cast, but will not be modified

        LAMA_CUSPARSE_CALL(
                        hipsparseScsrgeam( CUDAContext_cusparseHandle,
                                        numRows, numColumns,
                                        &alpha, descrCSR, nnzA, aValues, aIA, aJA,
                                        &beta, descrCSR, nnzB, bValues, bIA, bJA,
                                        descrCSR, cValues, const_cast<IndexType*>( cIA ), cJA ),
                        "hipsparseScsrgeam" )

        // synchronization might be redundant

        LAMA_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "hipsparseScsrgeam" )
    }

    template<>
    void CUSparseCSRUtils::matrixAdd(
                    IndexType cJA[],
                    double cValues[],
                    const IndexType cIA[],
                    const IndexType numRows,
                    const IndexType numColumns,
                    bool diagonalProperty,
                    const double alpha,
                    const IndexType aIA[],
                    const IndexType aJA[],
                    const double aValues[],
                    const double beta,
                    const IndexType bIA[],
                    const IndexType bJA[],
                    const double bValues[] )
    {
        LAMA_REGION( "CUDA.CSR.matrixAdd" )

        SCAI_LOG_INFO( logger, "matrixAdd for " << numRows << "x" << numColumns << " matrix" )

        LAMA_CHECK_CUDA_ACCESS

        hipsparseMatDescr_t descrCSR;

        LAMA_CUSPARSE_CALL( hipsparseCreateMatDescr( &descrCSR ), "hipsparseCreateMatDescr" )

        hipsparseSetMatType( descrCSR, HIPSPARSE_MATRIX_TYPE_GENERAL );
        hipsparseSetMatIndexBase( descrCSR, HIPSPARSE_INDEX_BASE_ZERO );

        int nnzA = 0; // aIA[ numRows ]
        int nnzB = 0;// bIA[ numColumns ]

        // we have not passed the number of non-zero values for A, B, so copy it

        hipMemcpy( &nnzA, &aIA[numRows], sizeof( IndexType ), hipMemcpyDeviceToHost );
        hipMemcpy( &nnzB, &bIA[numRows], sizeof( IndexType ), hipMemcpyDeviceToHost );

        // cIA requires const_cast, but will not be modified

        LAMA_CUSPARSE_CALL(
                        hipsparseDcsrgeam( CUDAContext_cusparseHandle,
                                        numRows, numColumns,
                                        &alpha, descrCSR, nnzA, aValues, aIA, aJA,
                                        &beta, descrCSR, nnzB, bValues, bIA, bJA,
                                        descrCSR, cValues, const_cast<IndexType*>( cIA ), cJA ),
                        "hipsparseDcsrgeam" )

        // synchronization might be redundant

        LAMA_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "hipsparseDcsrgeam" )
    }

    /* ------------------------------------------------------------------------------------------------------------------ */
    /*                                             matrixMultiply                                                         */
    /* ------------------------------------------------------------------------------------------------------------------ */

    template<>
    void CUSparseCSRUtils::matrixMultiply(
                    const IndexType cIA[],
                    IndexType cJA[],
                    float cValues[],
                    const IndexType m,
                    const IndexType n,
                    const IndexType k,
                    const float alpha,
                    bool diagonalProperty,
                    const IndexType aIA[],
                    const IndexType aJA[],
                    const float aValues[],
                    const IndexType bIA[],
                    const IndexType bJA[],
                    const float bValues[] )
    {
        LAMA_REGION( "CUDA.CSR.matrixMultiply" )

        SCAI_LOG_INFO( logger, "matrixMultiply, result is " << m << "x" << n << " CSR storage" )

        LAMA_CHECK_CUDA_ACCESS

        hipsparseMatDescr_t descrCSR;

        LAMA_CUSPARSE_CALL( hipsparseCreateMatDescr( &descrCSR ), "hipsparseCreateMatDescr" )

        hipsparseSetMatType( descrCSR, HIPSPARSE_MATRIX_TYPE_GENERAL );
        hipsparseSetMatIndexBase( descrCSR, HIPSPARSE_INDEX_BASE_ZERO );

        int nnzA = 0; // aIA[ m ]
        int nnzB = 0;// bIA[ numColumns ]

        // we have not passed the number of non-zero values for A, B, so copy it

        hipMemcpy( &nnzA, &aIA[m], sizeof( IndexType ), hipMemcpyDeviceToHost );
        hipMemcpy( &nnzB, &bIA[k], sizeof( IndexType ), hipMemcpyDeviceToHost );

        LAMA_ASSERT_EQUAL_ERROR( static_cast<float>( 1 ), alpha );

        LAMA_CUSPARSE_CALL(
                        hipsparseScsrgemm( CUDAContext_cusparseHandle,
                                        HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                        m, n, k,
                                        descrCSR, nnzA, aValues, aIA, aJA,
                                        descrCSR, nnzB, bValues, bIA, bJA,
                                        descrCSR, cValues, cIA, cJA ),
                        "hipsparseScsrgemm" )

        // synchronization might be redundant d

        LAMA_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "csrSparseMatmulS" )
    }

    template<>
    void CUSparseCSRUtils::matrixMultiply(
                    const IndexType cIA[],
                    IndexType cJA[],
                    double cValues[],
                    const IndexType m,
                    const IndexType n,
                    const IndexType k,
                    const double alpha,
                    bool diagonalProperty,
                    const IndexType aIA[],
                    const IndexType aJA[],
                    const double aValues[],
                    const IndexType bIA[],
                    const IndexType bJA[],
                    const double bValues[] )
    {
        LAMA_REGION( "CUDA.CSR.matrixMultiply" )

        SCAI_LOG_INFO( logger, "matrixMultiply, result is " << m << "x" << n << " CSR storage" )

        LAMA_CHECK_CUDA_ACCESS

        hipsparseMatDescr_t descrCSR;

        LAMA_CUSPARSE_CALL( hipsparseCreateMatDescr( &descrCSR ), "hipsparseCreateMatDescr" )

        hipsparseSetMatType( descrCSR, HIPSPARSE_MATRIX_TYPE_GENERAL );
        hipsparseSetMatIndexBase( descrCSR, HIPSPARSE_INDEX_BASE_ZERO );

        int nnzA = 0; // aIA[ m ]
        int nnzB = 0;// bIA[ n ]

        // we have not passed the number of non-zero values for A, B, so copy it

        hipMemcpy( &nnzA, &aIA[m], sizeof( IndexType ), hipMemcpyDeviceToHost );
        hipMemcpy( &nnzB, &bIA[k], sizeof( IndexType ), hipMemcpyDeviceToHost );

        LAMA_ASSERT_EQUAL_ERROR( static_cast<double>( 1 ), alpha );

        LAMA_CUSPARSE_CALL(
                        hipsparseDcsrgemm( CUDAContext_cusparseHandle,
                                        HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                        m, n, k,
                                        descrCSR, nnzA, aValues, aIA, aJA,
                                        descrCSR, nnzB, bValues, bIA, bJA,
                                        descrCSR, cValues, cIA, cJA ),
                        "hipsparseDcsrgemm" )

        // synchronization might be redundant d

        LAMA_CUDA_RT_CALL( hipStreamSynchronize( 0 ), "csrSparseMatmulD" )
    }

    /* ------------------------------------------------------------------------------------------------------------------ */

    /* --------------------------------------------------------------------------- */
    /*     Template instantiations via registration routine                        */
    /* --------------------------------------------------------------------------- */

    void CUSparseCSRUtils::setInterface( CSRUtilsInterface& CSRUtils )
    {
        SCAI_LOG_INFO( logger, "set CSR routines for CUSparse in Interface" )

        bool useCUSparse = true;

        // using CUSparse for CSR might be disabled explicitly by environment variable

        Settings::getEnvironment( useCUSparse, "LAMA_USE_CUSPARSE" );

        if ( !useCUSparse )
        {
            return;
        }

        // REGISTER1: overwrites previous settings

        LAMA_INTERFACE_REGISTER1_T( CSRUtils, normalGEMV, float )
        LAMA_INTERFACE_REGISTER1_T( CSRUtils, normalGEMV, double )

        LAMA_INTERFACE_REGISTER1_T( CSRUtils, convertCSR2CSC, float )
        LAMA_INTERFACE_REGISTER1_T( CSRUtils, convertCSR2CSC, double )

        LAMA_INTERFACE_REGISTER1( CSRUtils, matrixAddSizes )
        LAMA_INTERFACE_REGISTER1( CSRUtils, matrixMultiplySizes )

        LAMA_INTERFACE_REGISTER1_T( CSRUtils, matrixAdd, float )
        LAMA_INTERFACE_REGISTER1_T( CSRUtils, matrixAdd, double )

        LAMA_INTERFACE_REGISTER1_T( CSRUtils, matrixMultiply, float )
        LAMA_INTERFACE_REGISTER1_T( CSRUtils, matrixMultiply, double )
    }

    /* --------------------------------------------------------------------------- */
    /*    Static registration of the Utils routines                                */
    /* --------------------------------------------------------------------------- */

    bool CUSparseCSRUtils::registerInterface()
    {
        LAMAInterface& interface = LAMAInterfaceRegistry::getRegistry().modifyInterface( Context::CUDA );
        setInterface( interface.CSRUtils );
        return true;
    }

    /* --------------------------------------------------------------------------- */
    /*    Static initialiazion at program start                                    */
    /* --------------------------------------------------------------------------- */

    bool CUSparseCSRUtils::initialized = registerInterface();

} // namespace lama
