
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////

using namespace std;

int main( int argc, char** argv)
{

    int deviceCount = 0;

    hipError_t error_id = hipGetDeviceCount(&deviceCount);

    if ( error_id != hipSuccess )
    {
        cerr << "hipGetDeviceCount failed" << endl;
        return -1;
    }

    int driverVersion = 0;
    int runtimeVersion = 0;

    hipDriverGetVersion( &driverVersion );
    hipRuntimeGetVersion( &runtimeVersion );

    cout << "CUDA Driver Version = " << driverVersion << endl;
    cout << "CUDA Runtime Version = " << runtimeVersion << endl;

    for (int dev = 0; dev < deviceCount; ++dev) 
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties( &deviceProp, dev );
        cout << "Device " << dev << ": " << deviceProp.name << endl;
        cout << "  total amount of global memory = " << deviceProp.totalGlobalMem << " Byte"
             << " = " << deviceProp.totalGlobalMem / ( 1024.0 * 1024.0 ) << " MByte" << endl;
    }
}
